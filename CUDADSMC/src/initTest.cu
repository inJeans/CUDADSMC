#include "hip/hip_runtime.h"
//
//  initTest.cu
//  CUDADSMC
//
//  Created by Christopher Watkins on 7/08/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "initialSystemParameters.cuh"
#include "cudaHelpers.cuh"
#include "setUp.cuh"

int main(int argc, const char * argv[])
{
#pragma mark - Set up CUDA device
	// Flush device (useful for profiling)
    hipDeviceReset();
	
	int maxDevice = 0;
	maxDevice = setMaxCUDADevice( );
	
	int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, maxDevice);
	
#pragma mark - Set up atom system
	
	hiprandStatePhilox4_32_10_t *rngStates;
	hipMalloc( (void **)&rngStates, numberOfAtoms*sizeof(hiprandStatePhilox4_32_10_t) );
	
	int blockSize;
	int minGridSize;
	
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
									   &blockSize,
									   initRNG,
									   0,
									   numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	
	initRNG<<<gridSize,blockSize>>>( rngStates, numberOfAtoms );
    
    double4 *d_pos;
    double4 *d_vel;
    
	hipMalloc( (void **)&d_pos, numberOfAtoms*sizeof(double4) );
    hipMalloc( (void **)&d_vel, numberOfAtoms*sizeof(double4) );
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
									   &blockSize,
									   generateInitialDist,
									   0,
									   numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    
    generateInitialDist<<<gridSize,blockSize>>>( d_pos,
												d_vel,
												numberOfAtoms,
												Tinit,
												dBdz,
												rngStates );
    
    printf("gridSize = %i, blockSize = %i\n", gridSize, blockSize);
	
	double4 *h_pos = (*double4) calloc( numberOfAtoms, sizeof(double4) );
	double4 *h_vel = (*double4) calloc( numberOfAtoms, sizeof(double4) );
	
	hipMemcpy( h_pos, d_pos, numberOfAtoms*sizeof(double4), hipMemcpyDeviceToHost );
	hipMemcpy( h_vel, d_vel, numberOfAtoms*sizeof(double4), hipMemcpyDeviceToHost );
	
	
    // insert code here...
    printf("Hello, World!\n");
    return 0;
}
