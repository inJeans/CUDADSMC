#include "hip/hip_runtime.h"
//
//  main.c
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

#include "declareInitialSystemParameters.cuh"
#include "initialSystemParameters.cuh"
#include "cudaHelpers.cuh"
#include "hdf5Helpers.cuh"
#include "setUp.cuh"
#include "moveAtoms.cuh"
#include "collisions.cuh"

char filename[] = "motionTest.h5";
char groupname[] = "/atomData";

int main(int argc, const char * argv[])
{
#pragma mark - Set up CUDA device
	// Flush device (useful for profiling)
    hipDeviceReset();
	
	int maxDevice = 0;
	maxDevice = setMaxCUDADevice( );
	
	int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, maxDevice);
	
#pragma mark - Set up atom system
	
	if( argc == 2 )
	{
		dt = atof(argv[1]);
		loopsPerCollision = 0.3 / dt;
		printf("dt = %g\n", dt);
	}
	else if( argc > 2 )
	{
		printf("Too many arguments supplied.\n");
		return 0;
	}
	else
	{
		dt = 1.e-6;
	}
	
	copyConstantsToDevice<<<1,1>>>( dt );
	
	int sizeOfRNG = numberOfAtoms;
    
	hiprandStatePhilox4_32_10_t *d_rngStates;
	hipMalloc( (void **)&d_rngStates, sizeOfRNG*sizeof(hiprandStatePhilox4_32_10_t) );
	
	int blockSize;
	int minGridSize;
	
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                       &blockSize,
                                       (const void *) initRNG,
                                       0,
                                       sizeOfRNG );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	printf("initRNG:             gridSize = %i, blockSize = %i\n", gridSize, blockSize);
	initRNG<<<gridSize,blockSize>>>( d_rngStates, sizeOfRNG );
    
#pragma mark - Memory Allocation
    
    double time = 0;
    
    double3 *d_pos;
    double3 *d_vel;
    double3 *d_acc;
    
    cudaCalloc( (void **)&d_pos, numberOfAtoms, sizeof(double3) );
    cudaCalloc( (void **)&d_vel, numberOfAtoms, sizeof(double3) );
    cudaCalloc( (void **)&d_acc, numberOfAtoms, sizeof(double3) );
    
    double3 *h_pos = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    double3 *h_vel = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) generateInitialDist,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    printf("generateInitialDist: gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    generateInitialDist<<<gridSize,blockSize>>>( d_pos,
                                                 d_vel,
                                                 d_acc,
                                                 numberOfAtoms,
                                                 Tinit,
                                                 dBdz,
                                                 d_rngStates );
    
#pragma mark - Write Initial State
    
    createHDF5File( filename,
                    groupname );
    
    hipMemcpy( h_pos,
                d_pos,
                numberOfAtoms*sizeof(double3),
                hipMemcpyDeviceToHost );
	hipMemcpy( h_pos, d_pos, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
	char posDatasetName[] = "/atomData/positions";
    int3 atomDims = { numberOfAtoms, 3, 1 };
    hdf5FileHandle hdf5handlePos = createHDF5Handle( atomDims,
                                                     H5T_NATIVE_DOUBLE,
                                                     posDatasetName );
	intialiseHDF5File( hdf5handlePos,
                       filename );
	writeHDF5File( hdf5handlePos,
                   filename,
                   h_pos );
    
    hipMemcpy( h_vel, d_vel, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
    char velDatasetName[] = "/atomData/velocities";
    hdf5FileHandle hdf5handleVel = createHDF5Handle( atomDims,
                                                     H5T_NATIVE_DOUBLE,
                                                     velDatasetName );
    intialiseHDF5File( hdf5handleVel,
                       filename );
	writeHDF5File( hdf5handleVel,
                   filename,
                   h_vel );
    
    char timeDatasetName[] = "/atomData/simuatedTime";
    int3 timeDims = { 1, 1, 1 };
    hdf5FileHandle hdf5handleTime = createHDF5Handle( timeDims,
                                                      H5T_NATIVE_DOUBLE,
                                                      timeDatasetName );
    intialiseHDF5File( hdf5handleTime,
                       filename );
	writeHDF5File( hdf5handleTime,
                   filename,
                   &time );
    
#pragma mark - Main Loop
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) moveAtoms,
                                        0,
                                        numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    printf("moveAtoms:           gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
    for (int i=0; i<numberOfPrints; i++)
    {
        moveAtoms<<<gridSize,blockSize>>>( d_pos,
                                           d_vel,
                                           d_acc,
                                           numberOfAtoms );
        
        time += loopsPerCollision * dt;
        
        hipMemcpy( h_pos,
                    d_pos,
                    numberOfAtoms*sizeof(double3),
                    hipMemcpyDeviceToHost );
        hipMemcpy( h_vel,
                    d_vel,
                    numberOfAtoms*sizeof(double3),
                    hipMemcpyDeviceToHost );
        
        writeHDF5File( hdf5handlePos,
                       filename,
                       h_pos );
        writeHDF5File( hdf5handleVel,
                       filename,
                       h_vel );
        writeHDF5File( hdf5handleTime,
                       filename,
                       &time );
        
        if ((i/numberOfPrints*100)%5==0) {
            printf("...%f%% complete\n", (float)i/numberOfPrints*100.);
        }
    }
    
    // insert code here...
    printf("\n");
    
    free( h_pos );
    free( h_vel );
    
    hipFree( d_pos );
    hipFree( d_vel );
    hipFree( d_acc );
    
    hipDeviceReset();
    
    return 0;
}

