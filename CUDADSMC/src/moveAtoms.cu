#include "hip/hip_runtime.h"
//
//  moveAtoms.cu
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "declareDeviceSystemParameters.cuh"
#include "moveAtoms.cuh"
#include "vectorMath.cuh"
#include "math.h"

__global__ void copyConstantsToDevice( double dt )
{
	d_dt = dt;
	d_loopsPerCollision = 0.01 / d_dt;
	
	return;
}

__global__ void moveAtoms( double3 *pos, double3 *vel, double3 *acc, int numberOfAtoms )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
		double3 l_pos = pos[atom];
        double3 l_vel = vel[atom];
        double3 l_acc = acc[atom];
		
        for (int i=0; i<d_loopsPerCollision; i++) {
            velocityVerletUpdate( &l_pos,
                                  &l_vel,
                                  &l_acc );
        }
    
        pos[atom] = l_pos;
        vel[atom] = l_vel;
        acc[atom] = l_acc;
		
    }
    
    return;
}

__device__ void velocityVerletUpdate( double3 *pos, double3 *vel, double3 *acc )
{
    vel[0] = updateVelHalfStep( vel[0], acc[0] );
    pos[0] = updatePos( pos[0], vel[0] );
    acc[0] = updateAcc( pos[0] );
    vel[0] = updateVelHalfStep( vel[0], acc[0] );
}

__device__ void symplecticEulerUpdate( double3 *pos, double3 *vel, double3 *acc )
{
    acc[0] = updateAcc( pos[0] );
    vel[0] = updateVel( vel[0], acc[0] );
    pos[0] = updatePos( pos[0], vel[0] );
}

__device__ double3 updateVel( double3 vel, double3 acc )
{
    return vel + acc * d_dt;
}

__device__ double3 updateVelHalfStep( double3 vel, double3 acc )
{
    return vel + 0.5 * acc * d_dt;
}

__device__ double3 updatePos( double3 pos, double3 vel )
{
    return pos + vel * d_dt;
}

__device__ double3 updateAcc( double3 pos )
{
    double3 accel = make_double3( 0., 0., 0. );
    
    // The rsqrt function returns the reciprocal square root of its argument
	double potential = -0.5*d_gs*d_muB*d_dBdz*rsqrt(pos.x*pos.x + pos.y*pos.y + 4.0*pos.z*pos.z)/d_mRb;
	
	accel.x =       potential * pos.x;
	accel.y =       potential * pos.y;
	accel.z = 4.0 * potential * pos.z;
    
    return accel;
}