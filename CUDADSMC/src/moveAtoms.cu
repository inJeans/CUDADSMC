#include "hip/hip_runtime.h"
//
//  moveAtoms.cu
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "deviceSystemParameters.cuh"
#include "moveAtoms.cuh"
#include "vectorMath.cuh"
#include "math.h"

__global__ void copyConstantsToDevice( double dt )
{
    d_dt = dt;
    d_loopsPerCollision = 0.0007 / d_dt;
    
    return;
}

__global__ void moveAtoms( double3 *pos, double3 *vel, double3 *acc, int *atomID, int numberOfAtoms )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
         atom < numberOfAtoms;
         atom += blockDim.x * gridDim.x)
    {
        double3 l_pos = pos[atomID[atom]];
        double3 l_vel = vel[atomID[atom]];
        double3 l_acc = acc[atomID[atom]];
        
        //        for (int i=0; i<d_loopsPerCollision; i++) {
        velocityVerletUpdate(&l_pos,
                             &l_vel,
                             &l_acc );
        //        }
        
        pos[atomID[atom]] = l_pos;
        vel[atomID[atom]] = l_vel;
        acc[atomID[atom]] = l_acc;
        
    }
    
    return;
}

__device__ void velocityVerletUpdate( double3 *pos, double3 *vel, double3 *acc )
{
    vel[0] = updateVelHalfStep( pos[0], vel[0], acc[0] );
    pos[0] = updatePos( pos[0], vel[0] );
    acc[0] = updateAcc( pos[0] );
    vel[0] = updateVelHalfStep( pos[0], vel[0], acc[0] );
    
    return;
}

__device__ void symplecticEulerUpdate( double3 *pos, double3 *vel, double3 *acc )
{
    acc[0] = updateAcc( pos[0] );
    vel[0] = updateVel( pos[0], vel[0], acc[0] );
    pos[0] = updatePos( pos[0], vel[0] );
}

__device__ void explicitEulerUpdate( double3 *pos, double3 *vel, double3 *acc )
{
    double3 newAcc = updateAcc( pos[0] );
    double3 newVel = updateVel( pos[0], vel[0], newAcc );
    double3 newPos = updatePos( pos[0], vel[0] );
    
    pos[0] = newPos;
    vel[0] = newVel;
    acc[0] = newAcc;
}

__device__ double3 updateVel( double3 pos, double3 vel, double3 acc )
{
    return vel + acc * d_dt;
}

__device__ double3 updateVelHalfStep( double3 pos, double3 vel, double3 acc )
{
    return vel + 0.5 * acc * d_dt;
}

__device__ double3 updatePos( double3 pos, double3 vel )
{
    return pos + vel * d_dt;
}

__device__ double3 updateAcc( double3 pos )
{
    double3 accel = make_double3( 0., 0., 0. );
    
    double d2Bdr2 = d_dBdx*d_dBdx / d_B0 - 0.5 * d_d2Bdx2;
    double potential = -0.5 * d_gs * d_muB * d2Bdr2 / d_mRb;
    
    accel.x =       potential * pos.x;
    accel.y =       potential * pos.y;
    accel.z = 2.0 * potential * pos.z;
    
    return accel;
}