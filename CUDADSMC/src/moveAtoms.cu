#include "hip/hip_runtime.h"
//
//  moveAtoms.cu
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "deviceSystemParameters.cuh"
#include "moveAtoms.cuh"
#include "vectorMath.cuh"
#include "math.h"

__global__ void copyConstantsToDevice( double dt )
{
	d_dt = dt;
	d_loopsPerCollision = 0.0007 / d_dt;
	
	return;
}

__global__ void moveAtoms( double3 *pos, double3 *vel, double3 *acc, int numberOfAtoms )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
		double3 l_pos = pos[atom];
        double3 l_vel = vel[atom];
        double3 l_acc = acc[atom];
		
//        for (int i=0; i<d_loopsPerCollision; i++) {
            velocityVerletUpdate( &l_pos,
                                  &l_vel,
                                  &l_acc );
//        }
    
        pos[atom] = l_pos;
        vel[atom] = l_vel;
        acc[atom] = l_acc;
		
    }
    
    return;
}

__device__ void velocityVerletUpdate( double3 *pos, double3 *vel, double3 *acc )
{
    vel[0] = updateVelHalfStep( pos[0], vel[0], acc[0] );
    pos[0] = updatePos( pos[0], vel[0] );
    acc[0] = updateAcc( pos[0] );
    vel[0] = updateVelHalfStep( pos[0], vel[0], acc[0] );
    
    return;
}

__device__ void symplecticEulerUpdate( double3 *pos, double3 *vel, double3 *acc )
{
    acc[0] = updateAcc( pos[0] );
    vel[0] = updateVel( pos[0], vel[0], acc[0] );
    pos[0] = updatePos( pos[0], vel[0] );
}

__device__ double3 updateVel( double3 pos, double3 vel, double3 acc )
{
    return vel + acc * d_dt;
}

__device__ double3 updateVelHalfStep( double3 pos, double3 vel, double3 acc )
{
    return vel + 0.5 * acc * d_dt;
}

__device__ double3 updatePos( double3 pos, double3 vel )
{
    double3 newPos = pos + vel * d_dt;
    
    return newPos;
}

__device__ double3 updateAcc( double3 pos )
{
    double potential = -1.0 * d_gs * d_muB * d_dBdr / d_mRb;
    
    double3 accel = potential * pos;
    
    return accel;
}