#include "hip/hip_runtime.h"
//
//  moveAtoms.cu
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <hip/hip_runtime.h>

#include "deviceSystemParameters.cuh"
#include "moveAtoms.cuh"
#include "vectorMath.cuh"

__global__ void copyConstantsToDevice( double dt )
{
	d_dt = dt;
	
	return;
}

__global__ void moveAtoms( double3 *pos, double3 *vel, double3 *acc, hipDoubleComplex *psiUp, hipDoubleComplex *psiDn, int *atomID, int numberOfAtoms )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
        int l_atom = atomID[atom];
		double3 l_pos = pos[l_atom];
        double3 l_vel = vel[l_atom];
        double3 l_acc = acc[l_atom];
        hipDoubleComplex l_psiUp = psiUp[l_atom];
        hipDoubleComplex l_psiDn = psiDn[l_atom];
		
//        for (int i=0; i<d_loopsPerCollision; i++) {
            velocityVerletUpdate(&l_pos,
                                  &l_vel,
                                  &l_acc,
                                  &l_psiUp,
                                  &l_psiDn);
//        }
    
        pos[l_atom] = l_pos;
        vel[l_atom] = l_vel;
        acc[l_atom] = l_acc;
        psiUp[l_atom] = l_psiUp;
        psiDn[l_atom] = l_psiDn;
		
    }
    
    return;
}

__device__ void velocityVerletUpdate( double3 *pos, double3 *vel, double3 *acc, hipDoubleComplex *psiUp, hipDoubleComplex *psiDn )
{
    hipDoubleComplex psiUpTemp = psiUp[0];
    psiUp[0] = updatePsiUp(pos[0],
                           psiUp[0],
                           psiDn[0] );
    psiDn[0] = updatePsiDn(pos[0],
                           psiUpTemp,
                           psiDn[0] );
    
    vel[0]   = updateVelHalfStep(pos[0],
                                 vel[0],
                                 acc[0] );
    pos[0]   = updatePos(pos[0],
                         vel[0] );
    acc[0]   = updateAcc(pos[0],
                         psiUp[0],
                         psiDn[0] );
    vel[0]   = updateVelHalfStep(pos[0],
                                 vel[0],
                                 acc[0] );
    
    return;
}

__device__ void symplecticEulerUpdate( double3 *pos, double3 *vel, double3 *acc, hipDoubleComplex *psiUp, hipDoubleComplex *psiDn )
{
    hipDoubleComplex psiUpTemp = psiUp[0];
    psiUp[0] = updatePsiUp(pos[0],
                           psiUp[0],
                           psiDn[0] );
    psiDn[0] = updatePsiDn(pos[0],
                           psiUpTemp,
                           psiDn[0] );
    
    acc[0] = updateAcc(pos[0],
                       psiUp[0],
                       psiDn[0] );
    vel[0] = updateVel(pos[0],
                       vel[0],
                       acc[0] );
    pos[0]   = updatePos(pos[0],
                         vel[0] );
}

__device__ double3 updateVel( double3 pos, double3 vel, double3 acc )
{
    return vel + acc * d_dt;
}

__device__ double3 updateVelHalfStep( double3 pos, double3 vel, double3 acc )
{
    return vel + 0.5 * acc * d_dt;
}

__device__ double3 updatePos( double3 pos, double3 vel )
{
    double3 newPos = pos + vel * d_dt;
    
    return newPos;
}

__device__ double3 updateAcc( double3 pos, hipDoubleComplex psiUp, hipDoubleComplex psiDn )
{
    double3 accel = make_double3( 0., 0., 0. );
    
    double3 dBdx = diffMagneticFieldAlongx( pos );
    double3 dBdy = diffMagneticFieldAlongy( pos );
    double3 dBdz = diffMagneticFieldAlongz( pos );
    
    double potential = -1.0 * d_gs * d_muB / d_mRb;
    
    accel.x = potential * ( dBdx.x * (psiUp.x*psiDn.x + psiUp.y*psiDn.y) +
                            dBdx.y * (psiUp.x*psiDn.y - psiUp.y*psiDn.x) +
                            dBdx.z * (psiUp.x*psiUp.x + psiUp.y*psiUp.y - 0.5) );
    accel.y = potential * ( dBdy.x * (psiUp.x*psiDn.x + psiUp.y*psiDn.y) +
                            dBdy.y * (psiUp.x*psiDn.y - psiUp.y*psiDn.x) +
                            dBdy.z * (psiUp.x*psiUp.x + psiUp.y*psiUp.y - 0.5) );
    accel.z = potential * ( dBdz.x * (psiUp.x*psiDn.x + psiUp.y*psiDn.y) +
                            dBdz.y * (psiUp.x*psiDn.y - psiUp.y*psiDn.x) +
                            dBdz.z * (psiUp.x*psiUp.x + psiUp.y*psiUp.y - 0.5) );
//    accel.x = 0.5*potential * d_d2Bdx2 * pos.x;
//    accel.y = 0.5*potential * d_d2Bdx2 * pos.y;
//    accel.z = 0.5*potential * d_d2Bdx2 * pos.z;
    
    return accel;
}

__device__ hipDoubleComplex updatePsiUp(double3 pos,
                                       hipDoubleComplex psiUp,
                                       hipDoubleComplex psiDn )
{
    double3 Bn = getMagneticFieldNormal( pos );
    double  B  = getMagB( pos );
    
    double theta = 0.5 * d_gs * d_muB * B * d_dt / d_hbar;
    double sinTheta = sin(theta);
    double cosTheta = cos(theta);
    
    hipDoubleComplex newPsiUp = make_hipDoubleComplex( psiUp.x*cosTheta + ( Bn.x*psiDn.y - Bn.y*psiDn.x + Bn.z*psiUp.y)*sinTheta,
                                                     psiUp.y*cosTheta + (-Bn.x*psiDn.x - Bn.y*psiDn.y - Bn.z*psiUp.x)*sinTheta );
    
    return newPsiUp;
}

__device__ hipDoubleComplex updatePsiDn(double3 pos,
                                       hipDoubleComplex psiUp,
                                       hipDoubleComplex psiDn )
{
    double3 Bn = getMagneticFieldNormal( pos );
    double  B  = getMagB( pos );
    
    double theta = 0.5 * d_gs * d_muB * B * d_dt / d_hbar;
    double sinTheta = sin(theta);
    double cosTheta = cos(theta);
    
    hipDoubleComplex newPsiDn = make_hipDoubleComplex( psiDn.x*cosTheta + ( Bn.x*psiUp.y + Bn.y*psiUp.x - Bn.z*psiDn.y)*sinTheta,
                                                     psiDn.y*cosTheta + (-Bn.x*psiUp.x + Bn.y*psiUp.y + Bn.z*psiDn.x)*sinTheta );
    
    return newPsiDn;
}

__device__ double3 getMagneticFieldNormal( double3 pos )
{
    double3 B = getMagneticField( pos );
    
    double3 Bn = B / length( B );
    
    return Bn;
}

__device__ double getMagB( double3 pos )
{
    double3 B = getMagneticField( pos );
    
    return length( B );
}

__device__ double3 getMagneticField( double3 pos )
{
    double3 B = d_B0     * make_double3( 0., 0., 1. ) +
                d_dBdx   * make_double3( pos.x, -pos.y, 0. ) +
          0.5 * d_d2Bdx2 * make_double3( -pos.x*pos.z, -pos.y*pos.z, pos.z*pos.z - 0.5*(pos.x*pos.x+pos.y*pos.y) );
    
    return B;
}

__device__ double3 diffMagneticFieldAlongx( double3 pos )
{
    double3 dBdx = make_double3( d_dBdx - 0.5 * d_d2Bdx2 * pos.z,
                                 0.,
                                -0.5 * d_d2Bdx2 * pos.x );
    return dBdx;
}

__device__ double3 diffMagneticFieldAlongy( double3 pos )
{
    double3 dBdy = make_double3( 0.,
                                -d_dBdx - 0.5 * d_d2Bdx2 * pos.z,
                                -0.5 * d_d2Bdx2 * pos.y );
    return dBdy;
}

__device__ double3 diffMagneticFieldAlongz( double3 pos )
{
    double3 dBdz = make_double3(-0.5 * d_d2Bdx2 * pos.x,
                                -0.5 * d_d2Bdx2 * pos.y,
                                       d_d2Bdx2 * pos.z );
    return dBdz;
}