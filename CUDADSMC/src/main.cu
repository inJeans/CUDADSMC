#include "hip/hip_runtime.h"
//
//  main.c
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "declareInitialSystemParameters.cuh"
#include "initialSystemParameters.cuh"
#include "cudaHelpers.cuh"
#include "hdf5Helpers.cuh"
#include "setUp.cuh"
#include "moveAtoms.cuh"
#include "collisions.cuh"

char filename[] = "outputData.h5";
char groupname[] = "/atomData";

int main(int argc, const char * argv[])
{
#pragma mark - Set up CUDA device
	// Flush device (useful for profiling)
    hipDeviceReset();
	
	int maxDevice = 0;
	maxDevice = setMaxCUDADevice( );
	
	int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, maxDevice);
	
#pragma mark - Set up atom system
	
	hiprandStatePhilox4_32_10_t *rngStates;
	hipMalloc( (void **)&rngStates, numberOfAtoms*sizeof(hiprandStatePhilox4_32_10_t) );
	
	int blockSize;
	int minGridSize;
	
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        initRNG,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	printf("initRNG:             gridSize = %i, blockSize = %i\n", gridSize, blockSize);
	initRNG<<<gridSize,blockSize>>>( rngStates, numberOfAtoms );
    
#pragma mark - Memory Allocation
    
    double3 *d_pos;
    double3 *d_vel;
    double3 *d_acc;
    
    int2 *d_cellStartEnd;
    
    int *d_cellID;
    int *d_numberOfAtomsInCell;

    cudaCalloc( (void **)&d_pos, numberOfAtoms, sizeof(double3) );
    cudaCalloc( (void **)&d_vel, numberOfAtoms, sizeof(double3) );
    cudaCalloc( (void **)&d_acc, numberOfAtoms, sizeof(double3) );
    
    cudaCalloc( (void **)&d_cellStartEnd, numberOfCells+1, sizeof(int2) );
    
    cudaCalloc( (void **)&d_cellID, numberOfAtoms, sizeof(int) );
    cudaCalloc( (void **)&d_numberOfAtomsInCell, numberOfCells+1, sizeof(int) );
    
    double3 *h_pos = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    double3 *h_vel = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    
    int *h_numberOfAtomsInCell = (int*) calloc( numberOfCells+1, sizeof(int) );
	int *h_cellID = (int*) calloc( numberOfAtoms, sizeof(int) );
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        generateInitialDist,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    printf("generateInitialDist: gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    generateInitialDist<<<gridSize,blockSize>>>( d_pos,
                                                 d_vel,
                                                 d_acc,
                                                 numberOfAtoms,
                                                 Tinit,
                                                 dBdz,
                                                 rngStates );
    
    indexAtoms( d_pos,
                d_cellID );
	sortArrays( d_pos,
                d_vel,
                d_acc,
                d_cellID );
    
    createHDF5File( filename, groupname );
    
    hipMemcpy( h_pos, d_pos, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
	hdf5FileHandle hdf5handlePos = createHDF5Handle( numberOfAtoms, "/atomData/positions" );
	intialiseHDF5File( hdf5handlePos,
                       filename );
	writeHDF5File( hdf5handlePos,
                   filename,
                   h_pos );
    
    hipMemcpy( h_vel, d_vel, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
    hdf5FileHandle hdf5handleVel = createHDF5Handle( numberOfAtoms, "/atomData/velocities" );
	intialiseHDF5File( hdf5handleVel,
                       filename );
	writeHDF5File( hdf5handleVel,
                   filename,
                   h_vel );

#pragma mark - Main Loop
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                       &blockSize,
                                       moveAtoms,
                                       0,
                                       numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    printf("moveAtoms:           gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
    for (int i=0; i<1; i++)
    {
        indexAtoms( d_pos,
                    d_cellID );
        sortArrays( d_pos,
                    d_vel,
                    d_acc,
                    d_cellID );
		
		
        
		cellStartandEndKernel<<<gridSize,blockSize>>>( d_cellID, d_cellStartEnd, numberOfAtoms );
        findNumberOfAtomsInCell<<<numberOfCells+1,1>>>( d_cellStartEnd,
                                                         d_numberOfAtomsInCell,
                                                         numberOfCells );
        
//        collide<<<numberOfCells,64,6144>>>( d_pos,
//                                            d_vel,
//                                            d_cellID,
//                                            d_numberOfAtomsInCell,
//                                            numberOfCells );
        
        moveAtoms<<<gridSize,blockSize>>>( d_pos,
                                           d_vel,
                                           d_acc,
                                           numberOfAtoms );
    
        hipMemcpy( h_pos, d_pos, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
        hipMemcpy( h_vel, d_vel, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
    
        writeHDF5File( hdf5handlePos,
                       filename,
                       h_pos );
        writeHDF5File( hdf5handleVel,
                       filename,
                       h_vel );
        
        printf("i = %i\n", i);
    }
    // insert code here...
    printf("\n");
    
    free( h_pos );
    free( h_vel );
    free( h_numberOfAtomsInCell );
	free( h_cellID );
    
    hipFree( d_pos );
    hipFree( d_vel );
    hipFree( d_acc );
    hipFree( d_cellStartEnd );
    hipFree( d_cellID );
    hipFree( d_numberOfAtomsInCell );
    hipFree( rngStates );
    
    hipDeviceReset();
    
    return 0;
}

