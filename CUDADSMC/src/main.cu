#include "hip/hip_runtime.h"
//
//  main.c
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "initialSystemParameters.cuh"
#include "cudaHelpers.cuh"
#include "setUp.cuh"

int main(int argc, const char * argv[])
{
#pragma mark - Set up CUDA device
	// Flush device (useful for profiling)
    hipDeviceReset();
	
	int maxDevice = 0;
	maxDevice = setMaxCUDADevice( );
	
	int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, maxDevice);
	
#pragma mark - Set up atom system
	
	hiprandStatePhilox4_32_10_t *rngStates;
	hipMalloc( (void **)&rngStates, numberOfAtoms*sizeof(hiprandStatePhilox4_32_10_t) );
	
	int blockSize;
	int minGridSize;
	
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        initRNG,
                                        0,
                                        numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	
	initRNG<<<gridSize,blockSize>>>( rngStates, numberOfAtoms );
	
	printf("gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
    double4 *d_pos;
    double4 *d_vel;
    
	hipMalloc( (void **)&d_pos, numberOfAtoms*sizeof(double4) );
    hipMalloc( (void **)&d_vel, numberOfAtoms*sizeof(double4) );
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        generateInitialDist,
                                        0,
                                        numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    
    generateInitialDist<<<gridSize,blockSize>>>( d_pos,
                                                 d_vel,
                                                 numberOfAtoms,
                                                 Tinit,
                                                 dBdz,
                                                 rngStates );
    
    printf("gridSize = %i, blockSize = %i\n", gridSize, blockSize);
	
    // insert code here...
    printf("Hello, World!\n");
    return 0;
}

