#include "hip/hip_runtime.h"
//
//  main.c
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "initialSystemParameters.cuh"
#include "cudaHelpers.cuh"
#include "hdf5Helpers.cuh"
#include "setUp.cuh"
#include "moveAtoms.cuh"

char filename[] = "outputData.h5";

int main(int argc, const char * argv[])
{
#pragma mark - Set up CUDA device
	// Flush device (useful for profiling)
    hipDeviceReset();
	
	int maxDevice = 0;
	maxDevice = setMaxCUDADevice( );
	
	int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, maxDevice);
	
#pragma mark - Set up atom system
	
	hiprandStatePhilox4_32_10_t *rngStates;
	hipMalloc( (void **)&rngStates, numberOfAtoms*sizeof(hiprandStatePhilox4_32_10_t) );
	
	int blockSize;
	int minGridSize;
	
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        initRNG,
                                        0,
                                        numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	
	initRNG<<<gridSize,blockSize>>>( rngStates, numberOfAtoms );
	
	printf("gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
    double3 *d_pos;
    double3 *d_vel;
    double3 *d_acc;
    
	hipMalloc( (void **)&d_pos, numberOfAtoms*sizeof(double3) );
    hipMalloc( (void **)&d_vel, numberOfAtoms*sizeof(double3) );
    hipMalloc( (void **)&d_acc, numberOfAtoms*sizeof(double3) );
    
    hipMemset( d_pos, 0., numberOfAtoms*sizeof(double3) );
    hipMemset( d_vel, 0., numberOfAtoms*sizeof(double3) );
    hipMemset( d_acc, 0., numberOfAtoms*sizeof(double3) );
    
    double3 *h_pos = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    double3 *h_vel = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        generateInitialDist,
                                        0,
                                        numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    
    generateInitialDist<<<gridSize,blockSize>>>( d_pos,
                                                 d_vel,
                                                 d_acc,
                                                 numberOfAtoms,
                                                 Tinit,
                                                 dBdz,
                                                 rngStates );
    
    printf("gridSize = %i, blockSize = %i\n", gridSize, blockSize);
	
    createHDF5File( filename );
    
    hipMemcpy( h_pos, d_pos, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
	hdf5FileHandle hdf5handlePos = createHDF5Handle( numberOfAtoms, "/positions" );
	intialiseHDF5File( hdf5handlePos, filename );
	writeHDF5File( hdf5handlePos, filename, h_pos );
    
    hipMemcpy( h_vel, d_vel, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
    hdf5FileHandle hdf5handleVel = createHDF5Handle( numberOfAtoms, "/velocities" );
	intialiseHDF5File( hdf5handleVel, filename );
	writeHDF5File( hdf5handleVel, filename, h_vel );

#pragma mark - Moving atoms
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                       &blockSize,
                                       generateInitialDist,
                                       0,
                                       numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    printf("gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
    for (int i=0; i<5; i++)
    {
        moveAtoms<<<gridSize,blockSize>>>(d_pos,
                                          d_vel,
                                          d_acc,
                                          numberOfAtoms );
    
        hipMemcpy( h_pos, d_pos, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
        hipMemcpy( h_vel, d_vel, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
    
        writeHDF5File( hdf5handlePos, filename, h_pos );
        writeHDF5File( hdf5handleVel, filename, h_vel );
        
        printf("i = %i\n", i);
    }
    // insert code here...
    printf("\n");
    
    free( h_pos );
    
    hipFree( d_pos );
    hipFree( d_vel );
    hipFree( d_acc );
    hipFree( rngStates );
    
    hipDeviceReset();
    
    return 0;
}

