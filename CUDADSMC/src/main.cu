//
//  main.c
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "initialSystemParameters.cuh"
#include "cudaHelpers.cuh"
#include "setUp.cuh"

int main(int argc, const char * argv[])
{
#pragma mark - Set up CUDA device
	// Flush device (useful for profiling)
    hipDeviceReset();
	
	int maxDevice = 0;
	maxDevice = setMaxCUDADevice( );
	
	int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, maxDevice);
	
#pragma mark - Set up atom system
	
	hiprandStatePhilox4_32_10_t *rngStates;
	hipMalloc( (void **)&rngStates, numberOfAtoms*sizeof(hiprandStatePhilox4_32_10_t) );
	
	int blockSize;
	int minGridSize;
	
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, initRNG, 0, numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	
	initRNG<<<gridSize,blockSize>>>( rngStates, numberOfAtoms );
	
	printf("gridSize = %i,  lockSize = %i\n", gridSize, blockSize);
	
    // insert code here...
    printf("Hello, World!\n");
    return 0;
}

