#include "hip/hip_runtime.h"
//
//  main.c
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

#include "declareInitialSystemParameters.cuh"
#include "initialSystemParameters.cuh"
#include "cudaHelpers.cuh"
#include "hdf5Helpers.cuh"
#include "setUp.cuh"
#include "moveAtoms.cuh"
#include "collisions.cuh"

char filename[] = "outputData.h5";
char groupname[] = "/atomData";

int main(int argc, const char * argv[])
{
#pragma mark - Set up CUDA device
	// Flush device (useful for profiling)
    hipDeviceReset();
	
	int maxDevice = 0;
	maxDevice = setMaxCUDADevice( );
	
	int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, maxDevice);
	
#pragma mark - Set up atom system
	
	hiprandStatePhilox4_32_10_t *rngStates;
	hipMalloc( (void **)&rngStates, numberOfAtoms*sizeof(hiprandStatePhilox4_32_10_t) );
	
	int blockSize;
	int minGridSize;
	
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) initRNG,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	printf("initRNG:             gridSize = %i, blockSize = %i\n", gridSize, blockSize);
	initRNG<<<gridSize,blockSize>>>( rngStates, numberOfAtoms );
    
#pragma mark - Memory Allocation
    
    double3 *d_pos;
    double3 *d_vel;
    double3 *d_acc;
    
    float medianR;
    
    int2 *d_cellStartEnd;
    
    int *d_cellID;
    int *d_numberOfAtomsInCell;
    int *d_prefixScanNumberOfAtomsInCell;

    cudaCalloc( (void **)&d_pos, numberOfAtoms, sizeof(double3) );
    cudaCalloc( (void **)&d_vel, numberOfAtoms, sizeof(double3) );
    cudaCalloc( (void **)&d_acc, numberOfAtoms, sizeof(double3) );
    
    cudaCalloc( (void **)&d_cellStartEnd, numberOfCells+1, sizeof(int2) );
    
    cudaCalloc( (void **)&d_cellID, numberOfAtoms, sizeof(int) );
    cudaCalloc( (void **)&d_numberOfAtomsInCell, numberOfCells+1, sizeof(int) );
    cudaCalloc( (void **)&d_prefixScanNumberOfAtomsInCell, numberOfCells+1, sizeof(int) );
    
    double3 *h_pos = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    double3 *h_vel = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    
    thrust::device_ptr<int> th_numberOfAtomsInCell = thrust::device_pointer_cast( d_numberOfAtomsInCell );
    thrust::device_ptr<int> th_prefixScanNumberOfAtomsInCell = thrust::device_pointer_cast( d_prefixScanNumberOfAtomsInCell );
    
    int *h_numberOfAtomsInCell = (int*) calloc( numberOfCells+1, sizeof(int) );
	int *h_cellID = (int*) calloc( numberOfAtoms, sizeof(int) );
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) generateInitialDist,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    printf("generateInitialDist: gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    generateInitialDist<<<gridSize,blockSize>>>( d_pos,
                                                 d_vel,
                                                 d_acc,
                                                 numberOfAtoms,
                                                 Tinit,
                                                 dBdz,
                                                 rngStates );
    
    medianR = indexAtoms( d_pos,
                          d_cellID );
	sortArrays( d_pos,
                d_vel,
                d_acc,
                d_cellID );
    
    createHDF5File( filename, groupname );
    
    
    hipMemcpy( h_pos, d_pos, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
	char posDatasetName[] = "/atomData/positions";
    hdf5FileHandle hdf5handlePos = createHDF5Handle( numberOfAtoms,
                                                     posDatasetName );
	intialiseHDF5File( hdf5handlePos,
                       filename );
	writeHDF5File( hdf5handlePos,
                   filename,
                   h_pos );
    
    hipMemcpy( h_vel, d_vel, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
    char velDatasetName[] = "/atomData/velocities";
    hdf5FileHandle hdf5handleVel = createHDF5Handle( numberOfAtoms,
                                                     velDatasetName );
	intialiseHDF5File( hdf5handleVel,
                       filename );
	writeHDF5File( hdf5handleVel,
                   filename,
                   h_vel );
    
#pragma mark - Main Loop
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) moveAtoms,
                                        0,
                                        numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    printf("moveAtoms:           gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
    for (int i=0; i<10; i++)
    {
        medianR = indexAtoms( d_pos,
                              d_cellID );
        sortArrays( d_pos,
                    d_vel,
                    d_acc,
                    d_cellID );
		
		cellStartandEndKernel<<<gridSize,blockSize>>>( d_cellID,
                                                       d_cellStartEnd,
                                                       numberOfAtoms );
        findNumberOfAtomsInCell<<<numberOfCells+1,1>>>( d_cellStartEnd,
                                                        d_numberOfAtomsInCell,
                                                        numberOfCells );
        thrust::exclusive_scan( th_numberOfAtomsInCell,
                                th_numberOfAtomsInCell + numberOfCells + 1,
                                th_prefixScanNumberOfAtomsInCell );
        
        collide<<<numberOfCells,64>>>( d_pos,
                                       d_vel,
                                       d_prefixScanNumberOfAtomsInCell,
                                       medianR,
                                       numberOfCells );
        
        moveAtoms<<<gridSize,blockSize>>>( d_pos,
                                           d_vel,
                                           d_acc,
                                           numberOfAtoms );
    
        hipMemcpy( h_pos, d_pos, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
        hipMemcpy( h_vel, d_vel, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
    
        writeHDF5File( hdf5handlePos,
                       filename,
                       h_pos );
        writeHDF5File( hdf5handleVel,
                       filename,
                       h_vel );
        
        printf("i = %i\n", i);
    }
    // insert code here...
    printf("\n");
    
    free( h_pos );
    free( h_vel );
    free( h_numberOfAtomsInCell );
	free( h_cellID );
    
    hipFree( d_pos );
    hipFree( d_vel );
    hipFree( d_acc );
    hipFree( d_cellStartEnd );
    hipFree( d_cellID );
    hipFree( d_numberOfAtomsInCell );
    hipFree( d_prefixScanNumberOfAtomsInCell );
    hipFree( rngStates );
    
    hipDeviceReset();
    
    return 0;
}

