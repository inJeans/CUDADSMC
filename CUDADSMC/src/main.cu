#include "hip/hip_runtime.h"
//
//  main.c
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

#include "initialSystemParameters.cuh"
#include "cudaHelpers.cuh"
#include "setUp.cuh"
#include "moveAtoms.cuh"

int main(int argc, const char * argv[])
{
#pragma mark - Set up CUDA device
	// Flush device (useful for profiling)
    hipDeviceReset();
	
	int maxDevice = 0;
	maxDevice = setMaxCUDADevice( );
	
	int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, maxDevice);
	
#pragma mark - Set up atom system
	
	hiprandStatePhilox4_32_10_t *rngStates;
	hipMalloc( (void **)&rngStates, numberOfAtoms*sizeof(hiprandStatePhilox4_32_10_t) );
	
	int blockSize;
	int minGridSize;
	
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        initRNG,
                                        0,
                                        numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	
	initRNG<<<gridSize,blockSize>>>( rngStates, numberOfAtoms );
	
	printf("gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
    double4 *d_pos;
    double4 *d_vel;
    double4 *d_acc;
    
	hipMalloc( (void **)&d_pos, numberOfAtoms*sizeof(double4) );
    hipMalloc( (void **)&d_vel, numberOfAtoms*sizeof(double4) );
    hipMalloc( (void **)&d_acc, numberOfAtoms*sizeof(double4) );
    
    hipMemset( d_pos, 0., numberOfAtoms*sizeof(double4) );
    hipMemset( d_vel, 0., numberOfAtoms*sizeof(double4) );
    hipMemset( d_acc, 0., numberOfAtoms*sizeof(double4) );
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        generateInitialDist,
                                        0,
                                        numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    
    generateInitialDist<<<gridSize,blockSize>>>( d_pos,
                                                 d_vel,
                                                 d_acc,
                                                 numberOfAtoms,
                                                 Tinit,
                                                 dBdz,
                                                 rngStates );
    
    printf("gridSize = %i, blockSize = %i\n", gridSize, blockSize);
	
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                       &blockSize,
                                       generateInitialDist,
                                       0,
                                       numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    
    moveAtoms<<<gridSize,blockSize>>>( d_pos, d_vel, d_acc, numberOfAtoms );
    
    printf("gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
    // insert code here...
    printf("\n");
    
    hipFree( d_pos );
    hipFree( d_vel );
    hipFree( d_acc );
    hipFree( rngStates );
    
    hipDeviceReset();
    
    return 0;
}

