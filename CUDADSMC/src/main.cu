#include "hip/hip_runtime.h"
//
//  main.c
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "declareInitialSystemParameters.cuh"
#include "initialSystemParameters.cuh"
#include "cudaHelpers.cuh"
#include "hdf5Helpers.cuh"
#include "setUp.cuh"
#include "moveAtoms.cuh"
#include "collisions.cuh"

char filename[] = "outputData.h5";
char groupname[] = "/atomData";

int main(int argc, const char * argv[])
{
#pragma mark - Set up CUDA device
	// Flush device (useful for profiling)
    hipDeviceReset();
	
	int maxDevice = 0;
	maxDevice = setMaxCUDADevice( );
	
	int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, maxDevice);
	
#pragma mark - Set up atom system
	
	hiprandStatePhilox4_32_10_t *rngStates;
	hipMalloc( (void **)&rngStates, numberOfAtoms*sizeof(hiprandStatePhilox4_32_10_t) );
	
	int blockSize;
	int minGridSize;
	
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        initRNG,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	printf("initRNG:             gridSize = %i, blockSize = %i\n", gridSize, blockSize);
	initRNG<<<gridSize,blockSize>>>( rngStates, numberOfAtoms );
    
#pragma mark - Memory Allocation
    
    double3 *d_pos;
    double3 *d_vel;
    double3 *d_acc;
    
    int *d_cellID;

    cudaCalloc( (void **)&d_pos, numberOfAtoms, sizeof(double3) );
    cudaCalloc( (void **)&d_vel, numberOfAtoms, sizeof(double3) );
    cudaCalloc( (void **)&d_acc, numberOfAtoms, sizeof(double3) );
    
    cudaCalloc( (void **)&d_cellID, numberOfAtoms, sizeof(int) );
    
    double3 *h_pos = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    double3 *h_vel = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        generateInitialDist,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    printf("generateInitialDist: gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    generateInitialDist<<<gridSize,blockSize>>>( d_pos,
                                                 d_vel,
                                                 d_acc,
                                                 numberOfAtoms,
                                                 Tinit,
                                                 dBdz,
                                                 rngStates );
    
    indexAtoms( d_pos,
                d_cellID );
	sortArrays( d_pos,
                d_vel,
                d_acc,
                d_cellID );
    
    createHDF5File( filename, groupname );
    
    hipMemcpy( h_pos, d_pos, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
	hdf5FileHandle hdf5handlePos = createHDF5Handle( numberOfAtoms, "/atomData/positions" );
	intialiseHDF5File( hdf5handlePos,
                       filename );
	writeHDF5File( hdf5handlePos,
                   filename,
                   h_pos );
    
    hipMemcpy( h_vel, d_vel, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
    hdf5FileHandle hdf5handleVel = createHDF5Handle( numberOfAtoms, "/atomData/velocities" );
	intialiseHDF5File( hdf5handleVel,
                       filename );
	writeHDF5File( hdf5handleVel,
                   filename,
                   h_vel );

#pragma mark - Main Loop
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                       &blockSize,
                                       moveAtoms,
                                       0,
                                       numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    printf("moveAtoms:           gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
    for (int i=0; i<5; i++)
    {
        indexAtoms( d_pos,
                    d_cellID );
        sortArrays( d_pos,
                    d_vel,
                    d_acc,
                    d_cellID );
        
        moveAtoms<<<gridSize,blockSize>>>( d_pos,
                                           d_vel,
                                           d_acc,
                                           numberOfAtoms );
    
        hipMemcpy( h_pos, d_pos, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
        hipMemcpy( h_vel, d_vel, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
    
        writeHDF5File( hdf5handlePos,
                       filename,
                       h_pos );
        writeHDF5File( hdf5handleVel,
                       filename,
                       h_vel );
        
        printf("i = %i\n", i);
    }
    // insert code here...
    printf("\n");
    
    free( h_pos );
    
    hipFree( d_pos );
    hipFree( d_vel );
    hipFree( d_acc );
    hipFree( d_cellID );
    hipFree( rngStates );
    
    hipDeviceReset();
    
    return 0;
}

