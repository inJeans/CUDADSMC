#include "hip/hip_runtime.h"
//
//  main.c
//  CUDADSMC
//
//  Created by Christopher Watkins on 31/07/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/scan.h>

#include "declareInitialSystemParameters.cuh"
#include "initialSystemParameters.cuh"
//#include "declareDeviceSystemParameters.cuh"
#include "deviceSystemParameters.cuh"
#include "cudaHelpers.cuh"
#include "hdf5Helpers.cuh"
#include "vectorMath.cuh"
#include "setUp.cuh"
#include "moveAtoms.cuh"
#include "collisions.cuh"
#include "spinEvolution.cuh"
#include "evaporation.cuh"

struct isAligned
{
    __host__ __device__
    bool operator()(const hbool_t x)
    {
        return !x;
    }
};

char filename[] = "outputData.h5";
char groupname[] = "/atomData";

int main(int argc, const char * argv[])
{
#pragma mark - Set up CUDA device
	// Flush device (useful for profiling)
    hipDeviceReset();
	
	int maxDevice = 0;
	maxDevice = setMaxCUDADevice( );
	
	int numSMs;
    hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, maxDevice);
    
#pragma mark - Memory Allocation
    
    int sizeOfRNG = findRNGArrayLength( );
	
	hiprandStatePhilox4_32_10_t *d_rngStates;
	hipMalloc( (void **)&d_rngStates, sizeOfRNG*sizeof(hiprandStatePhilox4_32_10_t) );
    
    double3 *d_pos;
    double3 *d_vel;
    double3 *d_acc;
    
    double3 *d_flippedPos;
    double3 *d_flippedVel;
    
    zomplex *d_psiU;
    zomplex *d_psiD;
    
    double2 *d_oldPops2;
    
    double *d_sigvrmax;
    
    double time = 0.;
    double medianR;
    
    int2 *d_cellStartEnd;
    
    int *d_cellID;
    int *d_numberOfAtomsInCell;
    int *d_prefixScanNumberOfAtomsInCell;
    int *d_collisionCount;
    
    hbool_t *d_isSpinUp;

    cudaCalloc( (void **)&d_pos, numberOfAtoms, sizeof(double3) );
    cudaCalloc( (void **)&d_vel, numberOfAtoms, sizeof(double3) );
    cudaCalloc( (void **)&d_acc, numberOfAtoms, sizeof(double3) );
    
    cudaCalloc( (void **)&d_flippedPos, numberOfAtoms, sizeof(double3) );
    cudaCalloc( (void **)&d_flippedVel, numberOfAtoms, sizeof(double3) );
    
    cudaCalloc( (void **)&d_psiU, numberOfAtoms, sizeof(zomplex) );
    cudaCalloc( (void **)&d_psiD, numberOfAtoms, sizeof(zomplex) );
    
    cudaCalloc( (void **)&d_oldPops2, numberOfAtoms, sizeof(double2) );
    
    cudaCalloc( (void **)&d_sigvrmax, numberOfCells+1, sizeof(double) );
    
    cudaCalloc( (void **)&d_cellStartEnd, numberOfCells+1, sizeof(int2) );
    
    cudaCalloc( (void **)&d_cellID, numberOfAtoms, sizeof(int) );
    cudaCalloc( (void **)&d_numberOfAtomsInCell, numberOfCells+1, sizeof(int) );
    cudaCalloc( (void **)&d_prefixScanNumberOfAtomsInCell, numberOfCells+1, sizeof(int) );
    cudaCalloc( (void **)&d_collisionCount, numberOfCells+1, sizeof(int) );
    
    cudaCalloc( (void **)&d_isSpinUp, numberOfAtoms, sizeof(hbool_t) );
    
    double3 *h_pos = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    double3 *h_vel = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    
    double3 *h_flippedPos = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    double3 *h_flippedVel = (double3*) calloc( numberOfAtoms, sizeof(double3) );
    
    zomplex *h_psiU = (zomplex*) calloc( numberOfAtoms, sizeof(zomplex) );
    zomplex *h_psiD = (zomplex*) calloc( numberOfAtoms, sizeof(zomplex) );
    
    int *h_numberOfAtomsInCell = (int*) calloc( numberOfCells+1, sizeof(int) );
    int *h_collisionCount = (int*) calloc( numberOfCells+1, sizeof(int) );
	int *h_cellID = (int*) calloc( numberOfAtoms, sizeof(int) );
    
    hbool_t *h_isSpinUp = (hbool_t*) calloc( numberOfAtoms, sizeof(hbool_t) );
    
    thrust::device_ptr<int> th_numberOfAtomsInCell = thrust::device_pointer_cast( d_numberOfAtomsInCell );
    thrust::device_ptr<int> th_prefixScanNumberOfAtomsInCell = thrust::device_pointer_cast( d_prefixScanNumberOfAtomsInCell );
    
#pragma mark - Set up atom system
    
    if( argc == 2 )
	{
		dt = atof(argv[1]);
		printf("dt = %g\n", dt);
	}
	else if( argc > 2 )
	{
		printf("Too many arguments supplied.\n");
		return 0;
	}
	else
	{
		dt = 1.0e-6;
	}
	
    loopsPerCollision = 0.005 / dt;
    
	copyConstantsToDevice<<<1,1>>>( dt );
	
	int blockSize;
	int minGridSize;
	
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) initRNG,
                                        0,
                                        sizeOfRNG );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	printf("initRNG:             gridSize = %i, blockSize = %i\n", gridSize, blockSize);
	initRNG<<<gridSize,blockSize>>>( d_rngStates, sizeOfRNG );
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) generateInitialDist,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    printf("generateInitialDist: gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    generateInitialDist<<<gridSize,blockSize>>>( d_pos,
                                                 d_vel,
                                                 d_acc,
                                                 d_isSpinUp,
                                                 numberOfAtoms,
                                                 Tinit,
                                                 dBdz,
                                                 d_rngStates );
    
    setInitialWavefunction( d_psiU,
                            d_psiD,
                            d_oldPops2,
                            d_isSpinUp,
                            d_pos,
                            numberOfAtoms );
    
    initSigvrmax( d_sigvrmax, numberOfCells );
    
    medianR = indexAtoms( d_pos,
                          d_cellID );
    sortArrays( d_pos,
                d_vel,
                d_acc,
                d_psiU,
                d_psiD,
                d_oldPops2,
                d_isSpinUp,
                d_cellID );
    
#pragma mark - Write Initial State
    
    createHDF5File( filename,
                    groupname );
    
    hipMemcpy( h_pos,
                d_pos,
                numberOfAtoms*sizeof(double3),
                hipMemcpyDeviceToHost );
	char posDatasetName[] = "/atomData/positions";
    int3 atomDims = { numberOfAtoms, 3, 1 };
    hdf5FileHandle hdf5handlePos = createHDF5Handle( atomDims,
                                                     H5T_NATIVE_DOUBLE,
                                                     posDatasetName );
	intialiseHDF5File( hdf5handlePos,
                       filename );
	writeHDF5File( hdf5handlePos,
                   filename,
                   h_pos );
    
    hipMemcpy( h_vel,
                d_vel,
                numberOfAtoms*sizeof(double3),
                hipMemcpyDeviceToHost );
    char velDatasetName[] = "/atomData/velocities";
    hdf5FileHandle hdf5handleVel = createHDF5Handle( atomDims,
                                                     H5T_NATIVE_DOUBLE,
                                                     velDatasetName );
    intialiseHDF5File( hdf5handleVel,
                       filename );
	writeHDF5File( hdf5handleVel,
                   filename,
                   h_vel );
    
    hipMemcpy( h_flippedPos,
                d_flippedPos,
                numberOfAtoms*sizeof(double3),
                hipMemcpyDeviceToHost );
    char fPosDatasetName[] = "/atomData/flippedPos";
    hdf5FileHandle hdf5handlefPos = createHDF5Handle( atomDims,
                                                      H5T_NATIVE_DOUBLE,
                                                      fPosDatasetName );
    intialiseHDF5File( hdf5handlefPos,
                       filename );
    writeHDF5File( hdf5handlefPos,
                   filename,
                   h_flippedPos );
    
    hipMemcpy( h_flippedVel,
                d_flippedVel,
                numberOfAtoms*sizeof(double3),
                hipMemcpyDeviceToHost );
    char fVelDatasetName[] = "/atomData/flippedVel";
    hdf5FileHandle hdf5handlefVel = createHDF5Handle( atomDims,
                                                      H5T_NATIVE_DOUBLE,
                                                      fVelDatasetName );
    intialiseHDF5File( hdf5handlefVel,
                       filename );
    writeHDF5File( hdf5handlefVel,
                   filename,
                   h_flippedVel );
    
    hipMemcpy( h_psiU,
                d_psiU,
                numberOfAtoms*sizeof(zomplex),
                hipMemcpyDeviceToHost );
    char psiUDatasetName[] = "/atomData/psiU";
    int3 complexDims = { numberOfAtoms, 2, 1 };
    hdf5FileHandle hdf5handlePsiU = createHDF5Handle( complexDims,
                                                      H5T_NATIVE_DOUBLE,
                                                      psiUDatasetName );
    intialiseHDF5File( hdf5handlePsiU,
                       filename );
	writeHDF5File( hdf5handlePsiU,
                   filename,
                   h_psiU );
    
    hipMemcpy( h_psiD,
                d_psiD,
                numberOfAtoms*sizeof(zomplex),
                hipMemcpyDeviceToHost );
    char psiDDatasetName[] = "/atomData/psiD";
    hdf5FileHandle hdf5handlePsiD = createHDF5Handle( complexDims,
                                                      H5T_NATIVE_DOUBLE,
                                                      psiDDatasetName );
    intialiseHDF5File( hdf5handlePsiD,
                       filename );
	writeHDF5File( hdf5handlePsiD,
                   filename,
                   h_psiD );
    
    hipMemcpy( h_isSpinUp,
                d_isSpinUp,
                numberOfAtoms*sizeof(hbool_t),
                hipMemcpyDeviceToHost );
    char isSpinUpDatasetName[] = "/atomData/isSpinUp";
    int3 isSpinUpDims = { numberOfAtoms, 1, 1 };
    hdf5FileHandle hdf5handleIsSpinUp = createHDF5Handle( isSpinUpDims,
                                                          H5T_NATIVE_HBOOL,
                                                          isSpinUpDatasetName );
    intialiseHDF5File( hdf5handleIsSpinUp,
                       filename );
	writeHDF5File( hdf5handleIsSpinUp,
                   filename,
                   h_isSpinUp );
    
    hipMemcpy( h_collisionCount,
                d_collisionCount,
                (numberOfCells+1)*sizeof(int),
                hipMemcpyDeviceToHost );
	char collisionDatasetName[] = "/atomData/collisionCount";
    int3 collisionDims = { numberOfCells+1, 1, 1 };
    hdf5FileHandle hdf5handleCollision = createHDF5Handle( collisionDims,
                                                           H5T_NATIVE_INT,
                                                           collisionDatasetName );
	intialiseHDF5File( hdf5handleCollision,
                       filename );
	writeHDF5File( hdf5handleCollision,
                   filename,
                   h_collisionCount );
    
    hipMemcpy( h_numberOfAtomsInCell,
                d_numberOfAtomsInCell,
                (numberOfCells+1)*sizeof(int),
                hipMemcpyDeviceToHost );
	char nAtomDatasetName[] = "/atomData/atomCount";
    int3 nAtomDims = { numberOfCells+1, 1, 1 };
    hdf5FileHandle hdf5handlenAtom = createHDF5Handle( nAtomDims,
                                                       H5T_NATIVE_INT,
                                                       nAtomDatasetName );
	intialiseHDF5File( hdf5handlenAtom,
                       filename );
	writeHDF5File( hdf5handlenAtom,
                   filename,
                   h_numberOfAtomsInCell );
    
    char timeDatasetName[] = "/atomData/simuatedTime";
    int3 timeDims = { 1, 1, 1 };
    hdf5FileHandle hdf5handleTime = createHDF5Handle( timeDims,
                                                      H5T_NATIVE_DOUBLE,
                                                      timeDatasetName );
    intialiseHDF5File( hdf5handleTime,
                       filename );
	writeHDF5File( hdf5handleTime,
                   filename,
                   &time );
    
    char numberDatasetName[] = "/atomData/atomNumber";
    hdf5FileHandle hdf5handleNumber = createHDF5Handle( timeDims,
                                                      H5T_NATIVE_INT,
                                                      numberDatasetName );
    intialiseHDF5File( hdf5handleNumber,
                       filename );
    writeHDF5File( hdf5handleNumber,
                   filename,
                   &numberOfAtoms );
    
#pragma mark - Main Loop
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) moveAtoms,
                                        0,
                                        numberOfAtoms );
	
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    printf("moveAtoms:           gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
    for (int i=0; i<numberOfPrints; i++)
    {
#pragma mark Collide Atoms
        
//        medianR = indexAtoms( d_pos,
//                              d_cellID );
//        
//        sortArrays( d_pos,
//                    d_vel,
//                    d_acc,
//                    d_psiU,
//                    d_psiD,
//                    d_oldPops2,
//                    d_isSpinUp,
//                    d_cellID );
//		
//		deviceMemset<<<numberOfCells+1,1>>>( d_cellStartEnd,
//											 make_int2( -1, -1 ),
//											 numberOfCells + 1 );
//		cellStartandEndKernel<<<gridSize,blockSize>>>( d_cellID,
//                                                       d_cellStartEnd,
//                                                       numberOfAtoms );
//        findNumberOfAtomsInCell<<<numberOfCells+1,1>>>( d_cellStartEnd,
//                                                        d_numberOfAtomsInCell,
//                                                        numberOfCells );
//        thrust::exclusive_scan( th_numberOfAtomsInCell,
//                                th_numberOfAtomsInCell + numberOfCells + 1,
//                                th_prefixScanNumberOfAtomsInCell );
//        
//        collide<<<numberOfCells,1>>>( d_vel,
//                                       d_sigvrmax,
//                                       d_isSpinUp,
//                                       d_prefixScanNumberOfAtomsInCell,
//                                       d_collisionCount,
//                                       medianR,
//                                       numberOfCells,
//                                       d_rngStates,
//                                       d_cellID );
        
#pragma mark Evolve System
        
        for (int j=0; j<loopsPerCollision; j++) {
            
            unitaryEvolution<<<gridSize,blockSize>>>( d_psiU,
                                                      d_psiD,
                                                      d_oldPops2,
                                                      d_pos,
                                                      d_vel,
                                                      numberOfAtoms );
            
            moveAtoms<<<gridSize,blockSize>>>( d_pos,
                                               d_vel,
                                               d_acc,
                                               numberOfAtoms,
                                               d_isSpinUp );
            
            exponentialDecay<<<gridSize,blockSize>>>( d_psiU,
                                                      d_psiD,
                                                      d_pos,
                                                      d_isSpinUp,
                                                      numberOfAtoms );

            normaliseWavefunction<<<gridSize,blockSize>>>( d_psiU,
                                                           d_psiD,
                                                           numberOfAtoms );
        }
        
        projectSpins<<<gridSize,blockSize>>>( d_psiU,
                                             d_psiD,
                                             d_oldPops2,
                                             d_pos,
                                             d_vel,
                                             d_isSpinUp,
                                             d_rngStates,
                                             numberOfAtoms,
                                             d_flippedPos,
                                             d_flippedVel );
        
#pragma mark Evaoprate Atoms
        
        evaporateAtoms( d_pos,
                        d_vel,
                        d_acc,
                        d_psiU,
                        d_psiD,
                        d_oldPops2,
                        d_isSpinUp,
                        d_cellID,
                        medianR,
                        &numberOfAtoms );
        
        printf( "Number of atoms = %i, ", numberOfAtoms);
        
        time += loopsPerCollision * dt;
    
        hipMemcpy( h_pos, d_pos, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
        hipMemcpy( h_vel, d_vel, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
        hipMemcpy( h_psiU, d_psiU, numberOfAtoms*sizeof(zomplex), hipMemcpyDeviceToHost );
        hipMemcpy( h_psiD, d_psiD, numberOfAtoms*sizeof(zomplex), hipMemcpyDeviceToHost );
        hipMemcpy( h_isSpinUp, d_isSpinUp, numberOfAtoms*sizeof(hbool_t), hipMemcpyDeviceToHost );
        hipMemcpy( h_collisionCount, d_collisionCount, (numberOfCells+1)*sizeof(int), hipMemcpyDeviceToHost );
    
        writeHDF5File( hdf5handlePos,
                       filename,
                       h_pos );
        writeHDF5File( hdf5handleVel,
                       filename,
                       h_vel );
        writeHDF5File( hdf5handlePsiU,
                       filename,
                       h_psiU );
        writeHDF5File( hdf5handlePsiD,
                       filename,
                       h_psiD );
        writeHDF5File( hdf5handleIsSpinUp,
                       filename,
                       h_isSpinUp );
        writeHDF5File( hdf5handleCollision,
                       filename,
                       h_collisionCount );
        writeHDF5File( hdf5handleTime,
                       filename,
                       &time );
        writeHDF5File( hdf5handleNumber,
                       filename,
                       &numberOfAtoms );
        
        hipMemcpy( h_flippedPos, d_flippedPos, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
        hipMemcpy( h_flippedVel, d_flippedVel, numberOfAtoms*sizeof(double3), hipMemcpyDeviceToHost );
        
        writeHDF5File( hdf5handlefPos,
                       filename,
                       h_flippedPos );
        writeHDF5File( hdf5handlefVel,
                       filename,
                       h_flippedVel );
        
        printf("i = %i\n", i);
    }
    
    // insert code here...
    printf("\n");
    
    free( h_pos );
    free( h_vel );
    free( h_psiU );
    free( h_psiD );
    free( h_numberOfAtomsInCell );
	free( h_cellID );
    free( h_isSpinUp );
    
    free( h_flippedPos );
    free( h_flippedVel );
    
    hipFree( d_pos );
    hipFree( d_vel );
    hipFree( d_acc );
    hipFree( d_psiU );
    hipFree( d_psiD );
    hipFree( d_oldPops2 );
    hipFree( d_sigvrmax );
    hipFree( d_cellStartEnd );
    hipFree( d_cellID );
    hipFree( d_numberOfAtomsInCell );
    hipFree( d_prefixScanNumberOfAtomsInCell );
    hipFree( d_rngStates );
    hipFree( d_isSpinUp );
    
    hipFree( d_flippedPos );
    hipFree( d_flippedVel );
    
    hipDeviceReset();
    
    return 0;
}

