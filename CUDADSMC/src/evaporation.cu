#include "hip/hip_runtime.h"
//
//  evaporation.cu
//  CUDADSMC
//
//  Created by Christopher Watkins on 1/08/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include "vectorMath.cuh"
#include "evaporation.cuh"
#include "math.h"
#include "cudaHelpers.cuh"

#include "declareInitialSystemParameters.cuh"
#include "deviceSystemParameters.cuh"

void h_evaporationTag(double3 *d_pos,
                      double3 *d_vel,
                      double3 *d_evapPos,
                      double3 *d_evapVel,
                      hbool_t *d_atomIsSpinUp,
                      int     *d_atomID,
                      int     *d_evapTag,
                      double   Temp,
                      int      numberOfAtoms )
{
    int blockSize;
    int gridSize;
    
#ifdef CUDA65
    int minGridSize;
    
    hipOccupancyMaxPotentialBlockSize(&minGridSize,
                                       &blockSize,
                                       (const void *) evaporationTag,
                                       0,
                                       numberOfAtoms );
    gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
#else
    int device;
    hipGetDevice ( &device );
    int numSMs;
    hipDeviceGetAttribute(&numSMs,
                           hipDeviceAttributeMultiprocessorCount,
                           device);
    
    gridSize = 256*numSMs;
    blockSize = NUM_THREADS;
#endif
    
    evaporationTag<<<gridSize,blockSize>>>(d_pos,
                                           d_vel,
                                           d_evapPos,
                                           d_evapVel,
                                           d_atomIsSpinUp,
                                           d_atomID,
                                           d_evapTag,
                                           Temp,
                                           numberOfAtoms );
    
    return;
}

__global__ void evaporationTag(double3 *pos,
                               double3 *vel,
                               double3 *evapPos,
                               double3 *evapVel,
                               hbool_t *atomIsSpinUp,
                               int     *atomID,
                               int     *evapTag,
                               double   Temp,
                               int      numberOfAtoms )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
         atom < numberOfAtoms;
         atom += blockDim.x * gridDim.x)
    {
        int l_atom = atomID[atom];
        double3 l_pos = pos[l_atom];
        double3 l_vel = vel[l_atom];
        
        if ( atomIsSpinUp[l_atom] ) {
            evapTag[atom] = 0;
        }
        else
        {
            evapTag[atom] = 1;
            evapPos[l_atom] = l_pos;
            evapVel[l_atom] = l_vel;
        }
    }
    
    return;
}

double calculateTemp(double3 *d_vel,
                     int *d_atomID,
                     int numberOfAtoms )
{
    double *d_speed2;
    cudaCalloc( (void **)&d_speed2, numberOfAtoms, sizeof(double) );
    
    h_calculateSpeed2(d_vel,
                      d_atomID,
                      d_speed2,
                      numberOfAtoms );
    
    double T  = h_mRb / 3. / h_kB * findMean(d_speed2,
                                             numberOfAtoms );

//    printf("The temperature is %fuK\n", T * 1.e6 );
    
    hipFree( d_speed2 );
    
    return T;
}

void h_calculateSpeed2(double3 *d_vel,
                       int     *d_atomID,
                       double  *d_speed2,
                       int      numberOfAtoms )
{
    int blockSize;
    int gridSize;
    
#ifdef CUDA65
    int minGridSize;
    
    hipOccupancyMaxPotentialBlockSize(&minGridSize,
                                       &blockSize,
                                       (const void *) calculateSpeed2,
                                       0,
                                       numberOfAtoms );
    gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
#else
    int device;
    hipGetDevice ( &device );
    int numSMs;
    hipDeviceGetAttribute(&numSMs,
                           hipDeviceAttributeMultiprocessorCount,
                           device);
    
    gridSize = 256*numSMs;
    blockSize = NUM_THREADS;
#endif
    
    calculateSpeed2<<<gridSize,blockSize>>>(d_vel,
                                            d_atomID,
                                            d_speed2,
                                            numberOfAtoms );
    
    return;
}

__global__ void calculateSpeed2(double3 *vel,
                                int     *atomID,
                                double  *speed2,
                                int      numberOfAtoms )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
         atom < numberOfAtoms;
         atom += blockDim.x * gridDim.x)
    {
        speed2[atom] = dot( vel[atomID[atom]], vel[atomID[atom]] );
    }
    
    return;
}

double findMean( double *v, int N )
{
    thrust::device_ptr<double> th_v = thrust::device_pointer_cast( v );
    
    double sum = thrust::reduce( th_v, th_v + N );
    
    return sum / N;
}

__device__ double3 getMagneticFieldN( double3 pos )
{
    double3 B = getMagneticF( pos );
    
    double3 Bn = B / length( B );
    
    return Bn;
}

__device__ double3 getMagneticF( double3 pos )
{
    double3 B = d_dBdz * make_double3( 0.5*pos.x, 0.5*pos.y, -1.0*pos.z );
    
    return B;
}