#include "hip/hip_runtime.h"
//
//  evaporation.cu
//  CUDADSMC
//
//  Created by Christopher Watkins on 1/08/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//
#include <thrust/device_vector.h>
#include <thrust/reduce.h>

#include "vectorMath.cuh"
#include "evaporation.cuh"
#include "math.h"
#include "cudaHelpers.cuh"

#include "declareInitialSystemParameters.cuh"
#include "deviceSystemParameters.cuh"

void h_evaporationTag(double3 *d_pos,
                      double3 *d_vel,
                      double3 *d_evapPos,
                      double3 *d_evapVel,
                      hipDoubleComplex *d_psiUp,
                      hipDoubleComplex *d_psiDn,
                      int     *d_atomID,
                      int     *d_evapTag,
                      double   Temp,
                      int      numberOfAtoms )
{
    int blockSize;
    int gridSize;
    
#ifdef CUDA65
    int minGridSize;
    
    hipOccupancyMaxPotentialBlockSize(&minGridSize,
                                       &blockSize,
                                       (const void *) evaporationTag,
                                       0,
                                       sizeOfRNG );
    gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
#else
    int device;
    hipGetDevice ( &device );
    int numSMs;
    hipDeviceGetAttribute(&numSMs,
                           hipDeviceAttributeMultiprocessorCount,
                           device);
    
    gridSize = 256*numSMs;
    blockSize = NUM_THREADS;
#endif
    
    evaporationTag<<<gridSize,blockSize>>>(d_pos,
                                           d_vel,
                                           d_evapPos,
                                           d_evapVel,
                                           d_psiUp,
                                           d_psiDn,
                                           d_atomID,
                                           d_evapTag,
                                           Temp,
                                           numberOfAtoms );
    
    return;
}

__global__ void evaporationTag(double3 *pos,
                               double3 *vel,
                               double3 *evapPos,
                               double3 *evapVel,
                               hipDoubleComplex *psiUp,
                               hipDoubleComplex *psiDn,
                               int     *atomID,
                               int     *evapTag,
                               double   Temp,
                               int      numberOfAtoms )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
         atom < numberOfAtoms;
         atom += blockDim.x * gridDim.x)
    {
        int l_atom = atomID[atom];
        hipDoubleComplex l_psiUp = psiUp[l_atom];
        hipDoubleComplex l_psiDn = psiDn[l_atom];
        double3 l_pos = pos[l_atom];
        double3 l_vel = vel[l_atom];
        double3 Bn    = getMagneticFieldN( l_pos );
        
        double proj = 2. * Bn.x * ( l_psiUp.x*l_psiDn.x + l_psiUp.y*l_psiDn.y ) +
                      2. * Bn.y * ( l_psiUp.x*l_psiDn.y - l_psiUp.y*l_psiDn.x ) +
                      2. * Bn.z * ( l_psiUp.x*l_psiUp.x + l_psiUp.y*l_psiUp.y - 0.5 );
        
        if ( proj < 0.0 ) {
            evapTag[atom] = 1;
            evapPos[l_atom] = l_pos;
            evapVel[l_atom] = l_vel;
        }
        else
        {
            evapTag[atom] = 0;
        }
    }
    
    return;
}

double calculateTemp(double3 *d_vel,
                     int *d_atomID,
                     int numberOfAtoms )
{
    double *d_speed2;
    cudaCalloc( (void **)&d_speed2, numberOfAtoms, sizeof(double) );
    
    h_calculateSpeed2(d_vel,
                      d_atomID,
                      d_speed2,
                      numberOfAtoms );
    
    double T  = h_mRb / 3. / h_kB * findMean(d_speed2,
                                             numberOfAtoms );

//    printf("The temperature is %fuK\n", T * 1.e6 );
    
    hipFree( d_speed2 );
    
    return T;
}

void h_calculateSpeed2(double3 *d_vel,
                       int     *d_atomID,
                       double  *d_speed2,
                       int      numberOfAtoms )
{
    int blockSize;
    int gridSize;
    
#ifdef CUDA65
    int minGridSize;
    
    hipOccupancyMaxPotentialBlockSize(&minGridSize,
                                       &blockSize,
                                       (const void *) calculateSpeed2,
                                       0,
                                       sizeOfRNG );
    gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
#else
    int device;
    hipGetDevice ( &device );
    int numSMs;
    hipDeviceGetAttribute(&numSMs,
                           hipDeviceAttributeMultiprocessorCount,
                           device);
    
    gridSize = 256*numSMs;
    blockSize = NUM_THREADS;
#endif
    
    calculateSpeed2<<<gridSize,blockSize>>>(d_vel,
                                            d_atomID,
                                            d_speed2,
                                            numberOfAtoms );
    
    return;
}

__global__ void calculateSpeed2(double3 *vel,
                                int     *atomID,
                                double  *speed2,
                                int      numberOfAtoms )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
         atom < numberOfAtoms;
         atom += blockDim.x * gridDim.x)
    {
        speed2[atom] = dot( vel[atomID[atom]], vel[atomID[atom]] );
    }
    
    return;
}

double findMean( double *v, int N )
{
    thrust::device_ptr<double> th_v = thrust::device_pointer_cast( v );
    
    double sum = thrust::reduce( th_v, th_v + N );
    
    return sum / N;
}

__device__ double3 getMagneticFieldN( double3 pos )
{
    double3 B = getMagneticF( pos );
    
    double3 Bn = B / length( B );
    
    return Bn;
}

__device__ double3 getMagneticF( double3 pos )
{
    double3 B = d_dBdz * make_double3( 0.5 * pos.x, 0.5 * pos.y, -pos.z );
    
    return B;
}