#include "hip/hip_runtime.h"
//
//  collisions.cu
//  CUDADSMC
//
//  Created by Christopher Watkins on 19/08/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "declareInitialSystemParameters.cuh"
#include "deviceSystemParameters.cuh"
#include "collisions.cuh"
#include "cudaHelpers.cuh"

#pragma mark - Indexing

double indexAtoms( double3 *d_pos, int *d_cellID, int3 cellsPerDimension )
{
    double *d_radius;
    cudaCalloc( (void **)&d_radius, numberOfAtoms, sizeof(double) );
    
	h_calculateRadius( d_pos,
                       d_radius,
                       numberOfAtoms );
    
    double medianR = findMedian( d_radius,
                                 numberOfAtoms );
    
    printf("The median radius is %f\n", medianR );
    
    h_findAtomIndex( d_pos,
                     d_cellID,
                     medianR,
                     numberOfAtoms,
                     cellsPerDimension );
    
    hipFree( d_radius );
    
    return medianR;
}

void h_calculateRadius( double3 *d_pos, double *d_radius, int numberOfAtoms )
{
    int blockSize;
    int gridSize;
    
#ifdef CUDA65
    int minGridSize;
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) calculateRadius,
                                        0,
                                        sizeOfRNG );
    gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
#else
    int device;
    hipGetDevice ( &device );
    int numSMs;
    hipDeviceGetAttribute( &numSMs,
                            hipDeviceAttributeMultiprocessorCount,
                            device);
    
    gridSize = 256*numSMs;
    blockSize = NUM_THREADS;
#endif
    
    calculateRadius<<<gridSize,blockSize>>>( d_pos,
                                             d_radius,
                                             numberOfAtoms );
    
    return;
}

__global__ void calculateRadius( double3 *pos, double *radius, int numberOfAtoms )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
        radius[atom] = length( pos[atom] );
    }
    
    return;
}

double findMedian( double *v, int N )
{
    thrust::device_ptr<double> ptr = thrust::device_pointer_cast( v );
    
    thrust::sort( ptr,
                  ptr + N );
    
    double *d_median;
    cudaCalloc( (void **)&d_median, 1, sizeof(double) );
    
    getMedian<<<1,1>>>( v, d_median, N );
    
    double h_median;
    
    hipMemcpy( (void *)&h_median, d_median, 1*sizeof(double), hipMemcpyDeviceToHost );
    
    hipFree( d_median );
    
    return h_median;
}

__global__ void getMedian( double *v, double *median, int N)
{
    if (N % 2 == 0) {
        median[0] = 0.5*(v[N/2-1] + v[N/2]);
    }
    else {
        median[0] = v[(N-1)/2];
    }
    
    return;
}

void h_findAtomIndex( double3 *d_pos, int *d_cellID, double medianR, int numberOfAtoms, int3 cellsPerDimension )
{
    int blockSize;
    int gridSize;
    
#ifdef CUDA65
    int minGridSize;
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) calculateRadius,
                                        0,
                                        sizeOfRNG );
    gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
#else
    int device;
    hipGetDevice ( &device );
    int numSMs;
    hipDeviceGetAttribute( &numSMs,
                            hipDeviceAttributeMultiprocessorCount,
                            device);
    
    gridSize = 256*numSMs;
    blockSize = NUM_THREADS;
#endif
    
    findAtomIndex<<<gridSize,blockSize>>>( d_pos,
                                           d_cellID,
                                           medianR,
                                           numberOfAtoms,
                                           cellsPerDimension );
    
    return;
}

__global__ void findAtomIndex( double3 *pos, int *cellID, double medianR, int numberOfAtoms, int3 cellsPerDimension )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
        double3 l_pos = pos[atom];
        
        double3 gridMin    = getGridMin( medianR );
        double3 cellLength = getCellLength( medianR,
                                            cellsPerDimension );
    
        int3 cellIndices = getCellIndices( l_pos,
                                           gridMin,
                                           cellLength );
		
        cellID[atom] = getCellID( cellIndices, cellsPerDimension );
    }
    
    return;
}

__device__ double3 getCellLength( double medianR, int3 cellsPerDimension )
{
    double3 cellLength = 2.0 * d_maxGridWidth / cellsPerDimension;
    
    return cellLength;
}

__device__ int3 getCellIndices( double3 pos, double3 gridMin, double3 cellLength )
{
    int3 index = { 0, 0, 0 };
    
    index.x = __double2int_rd ( (pos.x - gridMin.x) / cellLength.x );
    index.y = __double2int_rd ( (pos.y - gridMin.y) / cellLength.y );
    index.z = __double2int_rd ( (pos.z - gridMin.z) / cellLength.z );
	
    return index;
}

__device__ int getCellID( int3 index, int3 cellsPerDimension )
{
    int cellID = 0;
    
    if (index.x > -1 && index.x < cellsPerDimension.x && index.y > -1 && index.y < cellsPerDimension.y && index.z > -1 && index.z < cellsPerDimension.z) {
        cellID = index.z*cellsPerDimension.x*cellsPerDimension.y + index.y*cellsPerDimension.x + index.x;
    }
    else {
        cellID = cellsPerDimension.x * cellsPerDimension.y * cellsPerDimension.z;
    }
    
    return cellID;
}

__device__ double3 getGridMin( double medianR )
{
    double3 gridMin = -1.0 * d_maxGridWidth;
    
    return  gridMin;
}

__global__ void cellStartandEndKernel( int *cellID, int2 *cellStartEnd, int numberOfAtoms )
{
	for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
        // Find the beginning of the cell
        if (atom == 0) {
            cellStartEnd[cellID[atom]].x = 0;
        }
        else if (cellID[atom] != cellID[atom-1]) {
            cellStartEnd[cellID[atom]].x = atom;
        }
        
        // Find the end of the cell
        if (atom == numberOfAtoms - 1) {
            cellStartEnd[cellID[atom]].y = numberOfAtoms-1;
        }
        else if (cellID[atom] != cellID[atom+1]) {
            cellStartEnd[cellID[atom]].y = atom;
        }
    }
    
    return;
}

__device__ void serialCellStartandEndKernel( int *cellID, int2 *cellStartEnd, int numberOfAtoms )
{
	for ( int atom = 0;
		  atom < numberOfAtoms;
		  atom++ )
	{
        // Find the beginning of the cell
        if (atom == 0) {
            cellStartEnd[cellID[atom]].x = 0;
        }
        else if (cellID[atom] != cellID[atom-1]) {
            cellStartEnd[cellID[atom]].x = atom;
        }
        
        // Find the end of the cell
        if (atom == numberOfAtoms - 1) {
            cellStartEnd[cellID[atom]].y = numberOfAtoms-1;
        }
        else if (cellID[atom] != cellID[atom+1]) {
            cellStartEnd[cellID[atom]].y = atom;
        }
    }
    
    return;
}

__global__ void findNumberOfAtomsInCell( int2 *cellStartEnd, int *numberOfAtomsInCell, int numberOfCells )
{
    for ( int cell = blockIdx.x * blockDim.x + threadIdx.x;
		  cell < numberOfCells+1;
		  cell += blockDim.x * gridDim.x)
	{
		if (cellStartEnd[cell].x == -1)
		{
			numberOfAtomsInCell[cell] = 0;
		}
		else
		{
			numberOfAtomsInCell[cell] = cellStartEnd[cell].y - cellStartEnd[cell].x + 1;
		}
    }
    
    return;
}

__device__ void serialFindNumberOfAtomsInCell( int2 *cellStartEnd, int *numberOfAtomsInCell, int numberOfCells )
{
    for (int cell = 0;
		 cell < numberOfCells;
		 cell++ )
	{
        numberOfAtomsInCell[cell] = cellStartEnd[cell].y - cellStartEnd[cell].x + 1;
    }
    
    return;
}

#pragma mark - Sorting

void sortArrays( double3 *d_pos,
                 double3 *d_vel,
                 double3 *d_acc,
                 int *d_cellID )
{
    thrust::device_ptr<double3> th_pos = thrust::device_pointer_cast( d_pos );
    thrust::device_ptr<double3> th_vel = thrust::device_pointer_cast( d_vel );
    thrust::device_ptr<double3> th_acc = thrust::device_pointer_cast( d_acc );
    
    thrust::device_ptr<int> th_cellID = thrust::device_pointer_cast( d_cellID );
    
    thrust::device_vector<int>  th_indices( numberOfAtoms );
    thrust::sequence( th_indices.begin(),
                      th_indices.end() );
    
    thrust::sort_by_key( th_cellID,
                         th_cellID + numberOfAtoms,
                         th_indices.begin() );
    
    double3 *d_sorted;
    cudaCalloc( (void **)&d_sorted, numberOfAtoms, sizeof(double3) );
    thrust::device_ptr<double3> th_sorted = thrust::device_pointer_cast( d_sorted );
    
    thrust::gather( th_indices.begin(),
                    th_indices.end(),
                    th_pos,
                    th_sorted );
    th_pos = th_sorted;

    thrust::gather( th_indices.begin(),
                    th_indices.end(),
                    th_vel,
                    th_sorted );
    th_vel = th_sorted;
    
    thrust::gather( th_indices.begin(),
                    th_indices.end(),
                    th_acc,
                    th_sorted );
    th_acc = th_sorted;
    
    hipFree( d_sorted );
    
    return;
}

#pragma mark - Collisions

__global__ void collide( double3 *vel,
                         double  *sigvrmax,
                         int     *prefixScanNumberOfAtomsInCell,
                         int     *collisionCount,
                         double   medianR,
                         double   alpha,
                         int3     cellsPerDimension,
                         int      numberOfCells,
                         hiprandStatePhilox4_32_10_t *rngState,
                         int *cellID )
{
    for ( int cell = blockIdx.x * blockDim.x + threadIdx.x;
          cell < numberOfCells;
          cell += blockDim.x * gridDim.x)
    {
        int numberOfAtomsInCell = prefixScanNumberOfAtomsInCell[cell+1] - prefixScanNumberOfAtomsInCell[cell];
        
        if (numberOfAtomsInCell > 2) {
            double3 cellLength = getCellLength( medianR,
                                                cellsPerDimension );
            
            d_dt = 1.0e-6;
            d_loopsPerCollision = 0.005 / d_dt;
            
            double cellVolume = cellLength.x * cellLength.y * cellLength.z;
            double Mc = 0.5 * (numberOfAtomsInCell - 1) * numberOfAtomsInCell;
            double lambda = ceil( Mc * alpha * d_loopsPerCollision * d_dt * sigvrmax[cell] / cellVolume ) / Mc;
            int Ncol = Mc*lambda;
            
            double3 velcm, newVel, pointOnSphere;
            
            double crossSection = 8.*d_pi*d_a*d_a;
            double magVrel;
            double ProbCol;
            
            for ( int l_collision = 0;
                  l_collision < Ncol;
                  l_collision++ )
            {
                hiprandStatePhilox4_32_10_t l_rngState = rngState[cell];
                
                int2 collidingAtoms = {0,0};
                
                if (numberOfAtomsInCell == 2) {
                    collidingAtoms.x = prefixScanNumberOfAtomsInCell[cell] + 0;
                    collidingAtoms.y = prefixScanNumberOfAtomsInCell[cell] + 1;
                }
                else {
                    collidingAtoms = prefixScanNumberOfAtomsInCell[cell] + chooseCollidingAtoms( numberOfAtomsInCell, &l_rngState );
                }
                
                magVrel = calculateRelativeVelocity( vel, collidingAtoms );
                
                // Check if this is the more probable than current most probable.
                if (magVrel*crossSection > sigvrmax[cell]) {
                    sigvrmax[cell] = magVrel * crossSection;
                }
                
                ProbCol = alpha * d_loopsPerCollision * d_dt / cellVolume * magVrel * crossSection / lambda;
                
                // Collide with the collision probability.
                if ( ProbCol > hiprand_uniform_double ( &l_rngState ) ) {
                    // Find centre of mass velocities.
                    velcm = 0.5*(vel[collidingAtoms.x] + vel[collidingAtoms.y]);
                    
                    // Generate a random velocity on the unit sphere.
                    pointOnSphere = getRandomPointOnSphere( &l_rngState );
                    newVel = magVrel * pointOnSphere;
                    
                    vel[collidingAtoms.x] = velcm - 0.5 * newVel;
                    vel[collidingAtoms.y] = velcm + 0.5 * newVel;
//                    collisionCount[cell] += d_alpha;
                    collisionCount[cell]++;
                }
                
                rngState[cell] = l_rngState;
                
            }
        }
    }

    return;
}

__device__ int2 chooseCollidingAtoms( int numberOfAtomsInCell, hiprandStatePhilox4_32_10_t *rngState )
{
    int2 collidingAtoms = { 0, 0 };
    
    // Randomly choose particles in this cell to collide.
    while (collidingAtoms.x == collidingAtoms.y) {
        collidingAtoms = double2Toint2_rd( hiprand_uniform2_double ( &rngState[0] ) * (numberOfAtomsInCell-1) );
    }
    
    return collidingAtoms;
}

__device__ double calculateRelativeVelocity( double3 *vel, int2 collidingAtoms )
{
    double3 vRel = vel[collidingAtoms.x] - vel[collidingAtoms.y];
    double magVrel = sqrt(vRel.x*vRel.x + vRel.y*vRel.y + vRel.z*vRel.z);
    
    return magVrel;
}

__device__ double3 getRandomPointOnSphere( hiprandStatePhilox4_32_10_t *rngState )
{
    double2 r1 = hiprand_normal2_double ( &rngState[0] );
    double  r2 = hiprand_normal_double  ( &rngState[0] );
    
    double3 pointOnSphere = make_double3( r1.x, r1.y, r2 ) * rsqrt( r1.x*r1.x + r1.y*r1.y + r2*r2 );
    
    return pointOnSphere;
}