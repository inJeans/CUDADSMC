#include "hip/hip_runtime.h"
//
//  collisions.cu
//  CUDADSMC
//
//  Created by Christopher Watkins on 19/08/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

#include "declareInitialSystemParameters.cuh"
#include "deviceSystemParameters.cuh"
#include "collisions.cuh"
#include "cudaHelpers.cuh"

#pragma mark - Indexing

double indexAtoms( double3 *d_pos, int *d_cellID )
{
    double *d_radius;
    cudaCalloc( (void **)&d_radius, numberOfAtoms, sizeof(double) );
    
    int blockSize;
	int minGridSize;
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) calculateRadius,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	printf("calculateRadius:     gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
	calculateRadius<<<gridSize,blockSize>>>( d_pos,
                                             d_radius,
                                             numberOfAtoms );
    
    double medianR = findMedian( d_radius,
                                 numberOfAtoms );
    
    printf("The median radius is %f\n", medianR );
    
    hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) findAtomIndex,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
	printf("findAtomIndex:       gridSize = %i, blockSize = %i\n", gridSize, blockSize);
    
    findAtomIndex<<<gridSize,blockSize>>>( d_pos, d_cellID, medianR, numberOfAtoms );
    
    hipFree( d_radius );
    
    return medianR;
}

__global__ void calculateRadius( double3 *pos, double *radius, int numberOfAtoms )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
        radius[atom] = length( pos[atom] );
    }
    
    return;
}

double findMedian( double *v, int N )
{
    thrust::device_ptr<double> ptr = thrust::device_pointer_cast( v );
    
    thrust::sort( ptr,
                  ptr + N );
    
    double *d_median;
    cudaCalloc( (void **)&d_median, 1, sizeof(double) );
    
    getMedian<<<1,1>>>( v, d_median, N );
    
    double h_median;
    
    hipMemcpy( (void *)&h_median, d_median, 1*sizeof(double), hipMemcpyDeviceToHost );
    
    hipFree( d_median );
    
    return h_median;
}

__global__ void getMedian( double *v, double *median, int N)
{
    if (N % 2 == 0) {
        median[0] = 0.5*(v[N/2-1] + v[N/2]);
    }
    else {
        median[0] = v[(N-1)/2];
    }
    
    return;
}

__global__ void findAtomIndex( double3 *pos, int *cellID, double medianR, int numberOfAtoms )
{
    for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
        double3 l_pos = pos[atom];
        
        double3 gridMin    = getGridMin( medianR );
        double3 cellLength = getCellLength( medianR );
    
        int3 cellIndices = getCellIndices( l_pos,
                                           gridMin,
                                           cellLength );
		
        cellID[atom] = getCellID( cellIndices, d_cellsPerDimension );
    }
    
    return;
}

__device__ double3 getCellLength( double medianR )
{
    double3 cellLength = 2.0 * d_meshWidth * medianR / d_cellsPerDimension;
    
    double3 maxLength = d_maxGridWidth / d_cellsPerDimension;
    
    if (cellLength.x > maxLength.x) {
        cellLength.x = maxLength.x;
    }
    if (cellLength.y > maxLength.y) {
        cellLength.y = maxLength.y;
    }
    if (cellLength.z > maxLength.z) {
        cellLength.z = maxLength.z;
    }
    
    return cellLength;
}

__device__ int3 getCellIndices( double3 pos, double3 gridMin, double3 cellLength )
{
    int3 index = { 0, 0, 0 };
    
    index.x = __double2int_rd ( (pos.x - gridMin.x) / cellLength.x );
    index.y = __double2int_rd ( (pos.y - gridMin.y) / cellLength.y );
    index.z = __double2int_rd ( (pos.z - gridMin.z) / cellLength.z );
	
    return index;
}

__device__ int getCellID( int3 index, int3 cellsPerDimension )
{
    int cellID = 0;
    
    if (index.x > -1 && index.x < cellsPerDimension.x && index.y > -1 && index.y < cellsPerDimension.y && index.z > -1 && index.z < cellsPerDimension.z) {
        cellID = index.z*cellsPerDimension.x*cellsPerDimension.y + index.y*cellsPerDimension.x + index.x;
    }
    else {
        cellID = cellsPerDimension.x * cellsPerDimension.y * cellsPerDimension.z;
    }
    
    return cellID;
}

__device__ double3 getGridMin( double medianR )
{
    double3 gridMin = -d_meshWidth * medianR * make_double3( 1., 1., 1. );
    
    return  gridMin;
}

__global__ void cellStartandEndKernel( int *cellID, int2 *cellStartEnd, int numberOfAtoms )
{
	for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
        // Find the beginning of the cell
        if (atom == 0) {
            cellStartEnd[cellID[atom]].x = 0;
        }
        else if (cellID[atom] != cellID[atom-1]) {
            cellStartEnd[cellID[atom]].x = atom;
        }
        
        // Find the end of the cell
        if (atom == numberOfAtoms - 1) {
            cellStartEnd[cellID[atom]].y = numberOfAtoms-1;
        }
        else if (cellID[atom] != cellID[atom+1]) {
            cellStartEnd[cellID[atom]].y = atom;
        }
    }
    
    return;
}

__device__ void serialCellStartandEndKernel( int *cellID, int2 *cellStartEnd, int numberOfAtoms )
{
	for (int atom = 0;
		 atom < numberOfAtoms;
		 atom++ )
	{
        // Find the beginning of the cell
        if (atom == 0) {
            cellStartEnd[cellID[atom]].x = 0;
        }
        else if (cellID[atom] != cellID[atom-1]) {
            cellStartEnd[cellID[atom]].x = atom;
        }
        
        // Find the end of the cell
        if (atom == numberOfAtoms - 1) {
            cellStartEnd[cellID[atom]].y = numberOfAtoms-1;
        }
        else if (cellID[atom] != cellID[atom+1]) {
            cellStartEnd[cellID[atom]].y = atom;
        }
    }
    
    return;
}

__global__ void findNumberOfAtomsInCell( int2 *cellStartEnd, int *numberOfAtomsInCell, int numberOfCells )
{
    for (int cell = blockIdx.x * blockDim.x + threadIdx.x;
		 cell < numberOfCells+1;
		 cell += blockDim.x * gridDim.x)
	{
		if (cellStartEnd[cell].x == -1)
		{
			numberOfAtomsInCell[cell] = 0;
		}
		else
		{
			numberOfAtomsInCell[cell] = cellStartEnd[cell].y - cellStartEnd[cell].x + 1;
		}
		if (numberOfAtomsInCell[cell] <  0) {
			printf("cell[%i] = %i\n", cell, numberOfAtomsInCell[cell]);
		}
    }
    
    return;
}

__device__ void serialFindNumberOfAtomsInCell( int2 *cellStartEnd, int *numberOfAtomsInCell, int numberOfCells )
{
    for (int cell = 0;
		 cell < numberOfCells;
		 cell++ )
	{
        numberOfAtomsInCell[cell] = cellStartEnd[cell].y - cellStartEnd[cell].x + 1;
    }
    
    return;
}

#pragma mark - Sorting

void sortArrays( double3 *d_pos,
                 double3 *d_vel,
                 double3 *d_acc,
                 zomplex *d_psiU,
                 zomplex *d_psiD,
                 double2 *d_oldPops2,
                 hbool_t *d_isSpinUp,
                 int *d_cellID )
{
    thrust::device_ptr<double3> th_pos = thrust::device_pointer_cast( d_pos );
    thrust::device_ptr<double3> th_vel = thrust::device_pointer_cast( d_vel );
    thrust::device_ptr<double3> th_acc = thrust::device_pointer_cast( d_acc );
    
    thrust::device_ptr<zomplex> th_psiU = thrust::device_pointer_cast( d_psiU );
    thrust::device_ptr<zomplex> th_psiD = thrust::device_pointer_cast( d_psiD );
    
    thrust::device_ptr<double2> th_oldPops2 = thrust::device_pointer_cast( d_oldPops2 );
    
    thrust::device_ptr<hbool_t> th_isSpinUp = thrust::device_pointer_cast( d_isSpinUp );
    
    thrust::device_ptr<int> th_cellID = thrust::device_pointer_cast( d_cellID );
    
    thrust::device_vector<int>  th_indices( numberOfAtoms );
    thrust::sequence( th_indices.begin(),
                      th_indices.end() );
    
    thrust::sort_by_key( th_cellID,
                         th_cellID + numberOfAtoms,
                         th_indices.begin() );
    
    double3 *d_sorted;
    cudaCalloc( (void **)&d_sorted, numberOfAtoms, sizeof(double3) );
    thrust::device_ptr<double3> th_sorted = thrust::device_pointer_cast( d_sorted );
    
    thrust::gather( th_indices.begin(),
                    th_indices.end(),
                    th_pos,
                    th_sorted );
    th_pos = th_sorted;

    thrust::gather( th_indices.begin(),
                    th_indices.end(),
                    th_vel,
                    th_sorted );
    th_vel = th_sorted;
    
    thrust::gather( th_indices.begin(),
                    th_indices.end(),
                    th_acc,
                    th_sorted );
    th_acc = th_sorted;
    
    hipFree( d_sorted );
    
    zomplex *d_sortedz;
    cudaCalloc( (void **)&d_sortedz, numberOfAtoms, sizeof(zomplex) );
    thrust::device_ptr<zomplex> th_sortedz = thrust::device_pointer_cast( d_sortedz );
    
    thrust::gather( th_indices.begin(),
                    th_indices.end(),
                    th_psiU,
                    th_sortedz );
    th_psiU = th_sortedz;
    
    thrust::gather( th_indices.begin(),
                    th_indices.end(),
                    th_psiD,
                    th_sortedz );
    th_psiD = th_sortedz;
    
    hipFree( d_sortedz );
    
    double2 *d_sorted2;
    cudaCalloc( (void **)&d_sorted2, numberOfAtoms, sizeof(double2) );
    thrust::device_ptr<double2> th_sorted2 = thrust::device_pointer_cast( d_sorted2 );
    
    thrust::gather( th_indices.begin(),
                    th_indices.end(),
                    th_oldPops2,
                    th_sorted2 );
    th_oldPops2 = th_sorted2;
    
    hipFree( d_sorted2 );
    
    hbool_t *d_sortedb;
    cudaCalloc( (void **)&d_sortedb, numberOfAtoms, sizeof(hbool_t) );
    thrust::device_ptr<hbool_t> th_sortedb = thrust::device_pointer_cast( d_sortedb );
    
    thrust::gather( th_indices.begin(),
                    th_indices.end(),
                    th_isSpinUp,
                    th_sortedb );
    th_isSpinUp = th_sortedb;
    
    hipFree( d_sortedb );
    
    return;
}

#pragma mark - Collisions

__global__ void collide( double3 *vel,
                         double  *sigvrmax,
                         hbool_t *isSpinUp,
                         int     *prefixScanNumberOfAtomsInCell,
                         int     *collisionCount,
                         double   medianR,
                         int      numberOfCells,
                         hiprandStatePhilox4_32_10_t *rngState,
                         int *cellID )
{
    int cell   = blockIdx.x;
    int numberOfAtomsInCell = prefixScanNumberOfAtomsInCell[cell+1] - prefixScanNumberOfAtomsInCell[cell];
    int g_atom = 0;
    
    double3 cellLength = getCellLength( medianR );
    
    d_dt = 1.0e-6;
	d_loopsPerCollision = 0.005 / d_dt;
    
    __shared__ double3 sh_vel[MAXATOMS];
    __syncthreads();
    
    if (numberOfAtomsInCell > MAXATOMS) {
        numberOfAtomsInCell = MAXATOMS;
    }
    
    for ( int l_atom = threadIdx.x;
          l_atom < numberOfAtomsInCell;
		  l_atom += blockDim.x )
    {
        g_atom = prefixScanNumberOfAtomsInCell[cell] + l_atom;
        
        sh_vel[l_atom] = vel[g_atom];
    }
    __syncthreads();
    
    double cellVolume = cellLength.x * cellLength.y * cellLength.z;
    int Mc = __double2int_ru( 0.5 * d_alpha * (numberOfAtomsInCell - 1) * numberOfAtomsInCell * d_loopsPerCollision * d_dt * sigvrmax[cell] / cellVolume );
//
//    int2 collidingAtoms, g_collidingAtoms;
//    
    double3 velcm, newVel, pointOnSphere;

    double crossSection = 8.*d_pi*d_a*d_a;
    double magVrel;
    double ProbCol;
    
    if (threadIdx.x==0) {
    for ( int l_collision = 0;
            l_collision < Mc;
            l_collision++ )
    {
        int g_collisionId =  l_collision%64 + cell*blockDim.x;
        hiprandStatePhilox4_32_10_t l_rngState = rngState[g_collisionId];
        
        int2 collidingAtoms = {0,0};
        
        if (numberOfAtomsInCell < 2) {
            return;
        }
        else if (numberOfAtomsInCell == 2) {
            collidingAtoms.x = 0;
            collidingAtoms.y = 1;
        }
        else {
            collidingAtoms = chooseCollidingAtoms( numberOfAtomsInCell, &l_rngState );
        }
        
        magVrel = calculateRelativeVelocity( sh_vel, collidingAtoms );
//        double minVel;
//        if (length(sh_vel[collidingAtoms.x]) < length(sh_vel[collidingAtoms.y])) {
//            minVel = length(sh_vel[collidingAtoms.x]);
//        }
//        else {
//             minVel = length( sh_vel[collidingAtoms.y]);
//        }
//        
//        if (magVrel / minVel > 5. )
//        {
//            printf("Woah massive velocity difference: %%%g, |v1| = %g, |v2| = %g\n", magVrel/minVel*100., length(sh_vel[collidingAtoms.x]), length(sh_vel[collidingAtoms.y]));
//        }

        // Check if this is the more probable than current most probable.
        if (magVrel*crossSection > sigvrmax[cell]) {
            sigvrmax[cell] = magVrel * crossSection;
        }

        ProbCol = 0.5 * d_alpha * d_loopsPerCollision * d_dt / cellVolume * magVrel * crossSection * numberOfAtomsInCell * ( numberOfAtomsInCell - 1. ) / Mc;

//        printf("Mc = %i, ProbCol = %g\n", Mc, ProbCol );
        
		// Collide with the collision probability.
        if ( ProbCol > hiprand_uniform_double ( &l_rngState ) ) {
            // Find centre of mass velocities.
            velcm = 0.5*(sh_vel[collidingAtoms.x] + sh_vel[collidingAtoms.y]);
            
            // Generate a random velocity on the unit sphere.
            pointOnSphere = getRandomPointOnSphere( &l_rngState );
            newVel = magVrel * pointOnSphere;
            
            sh_vel[collidingAtoms.x] = velcm - 0.5 * newVel;
            sh_vel[collidingAtoms.y] = velcm + 0.5 * newVel;
            
            //            atomicAdd( &collisionCount[cell], d_alpha );
            collisionCount[cell] += d_alpha;
        }
        
        rngState[g_collisionId] = l_rngState;
        
    }
    }
    __syncthreads();
    
    for ( int l_atom = threadIdx.x;
         l_atom < numberOfAtomsInCell;
         l_atom += blockDim.x )
    {
        g_atom = prefixScanNumberOfAtomsInCell[cell] + l_atom;
        
        vel[g_atom] = sh_vel[l_atom];
    }
    
    __syncthreads();
    
    return;
}

__device__ int2 chooseCollidingAtoms( int numberOfAtomsInCell, hiprandStatePhilox4_32_10_t *rngState )
{
    int2 collidingAtoms = { 0, 0 };
    
    // Randomly choose particles in this cell to collide.
    while (collidingAtoms.x == collidingAtoms.y) {
        collidingAtoms = double2Toint2_rd( hiprand_uniform2_double ( &rngState[0] ) * (numberOfAtomsInCell-1) );
    }
    
    return collidingAtoms;
}

__device__ double calculateRelativeVelocity( double3 *vel, int2 collidingAtoms )
{
    double3 vRel = vel[collidingAtoms.x] - vel[collidingAtoms.y];
    double magVrel = sqrt(vRel.x*vRel.x + vRel.y*vRel.y + vRel.z*vRel.z);
    
    return magVrel;
}

__device__ double3 getRandomPointOnSphere( hiprandStatePhilox4_32_10_t *rngState )
{
    double2 r1 = hiprand_normal2_double ( &rngState[0] );
    double  r2 = hiprand_normal_double  ( &rngState[0] );
    
    double3 pointOnSphere = make_double3( r1.x, r1.y, r2 ) * rsqrt( r1.x*r1.x + r1.y*r1.y + r2*r2 );
    
    return pointOnSphere;
}

__global__ void shmemTest( double3 *vel,
                          double3 *vout,
                          int     *prefixScanNumberOfAtomsInCell,
                          hiprandStatePhilox4_32_10_t *rngState )
{
    int cell   = blockIdx.x;
    int numberOfAtomsInCell = prefixScanNumberOfAtomsInCell[cell+1] - prefixScanNumberOfAtomsInCell[cell];
    int g_atom = 0;
    
    __shared__ double3 sh_vel[MAXATOMS];
    __syncthreads();
    
    if (numberOfAtomsInCell > MAXATOMS) {
        numberOfAtomsInCell = MAXATOMS;
    }
    
    for ( int l_atom = threadIdx.x;
         l_atom < numberOfAtomsInCell;
         l_atom += blockDim.x )
    {
        g_atom = prefixScanNumberOfAtomsInCell[cell] + l_atom;
        
        sh_vel[l_atom] = vel[g_atom];
    }
    __syncthreads();
    
    for ( int l_collision = threadIdx.x;
                 l_collision < 100;
                l_collision += blockDim.x )
    {
        int g_collisionId =  l_collision%64 + cell*blockDim.x;
        hiprandStatePhilox4_32_10_t l_rngState = rngState[g_collisionId];
        
        int2 collidingAtoms = {0,0};
        
        if (numberOfAtomsInCell < 2) {
            return;
        }
        else if (numberOfAtomsInCell == 2) {
            collidingAtoms.x = 0;
            collidingAtoms.y = 1;
        }
        else {
            collidingAtoms = chooseCollidingAtoms( numberOfAtomsInCell, &l_rngState );
        }
        
        int2 g_collidingAtoms = prefixScanNumberOfAtomsInCell[cell] + collidingAtoms;
        
        if (vel[g_collidingAtoms.x].x != sh_vel[collidingAtoms.x].x) {
            printf("g_vel[%i] = {%g, %g, %g}, sh_vel[%i] = {%g, %g, %g}, cell%i\n", g_collidingAtoms.x, vel[g_collidingAtoms.x].x, vel[g_collidingAtoms.x].y, vel[g_collidingAtoms.x].z, collidingAtoms.x, sh_vel[collidingAtoms.x].x, sh_vel[collidingAtoms.x].y, sh_vel[collidingAtoms.x].z, cell);
        }
        rngState[g_collisionId] = l_rngState;
        
    }
    
    
    __syncthreads();
    
    return;
}