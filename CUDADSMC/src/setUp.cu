#include "hip/hip_runtime.h"
//
//  setUp.cu
//  CUDADSMC
//
//  Created by Christopher Watkins on 1/08/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include "vectorMath.cuh"
#include "setUp.cuh"
#include "math.h"
#include "cudaHelpers.cuh"

#include "declareInitialSystemParameters.cuh"
#include "deviceSystemParameters.cuh"

#pragma mark - Random Number Generator
int findRNGArrayLength( void )
{
    int sizeOfRNG = 0;
    
    if (numberOfAtoms > 64*numberOfCells) {
		sizeOfRNG = numberOfAtoms;
	}
	else
	{
		sizeOfRNG = 64*numberOfCells;
	}
    
    return sizeOfRNG;
}

__global__ void initRNG( hiprandStatePhilox4_32_10_t *rngState, int numberOfAtoms )
{
	for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
		// Each thread gets the same seed, a different sequence
		// number and no offset
		hiprand_init( 1234, atom, 0, &rngState[atom] );
	}
	
	return;
}

#pragma mark - Initial Distribution

// Kernel to generate the initial distribution
__global__ void generateInitialDist(double3 *pos,
                                    double3 *vel,
                                    double3 *acc,
                                    hbool_t *isSpinUp,
                                    int      numberOfAtoms,
									double   Temp,
									double   dBdz,
									hiprandStatePhilox4_32_10_t *rngState) {
    
	for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
		/* Copy state to local memory for efficiency */
		hiprandStatePhilox4_32_10_t localrngState = rngState[atom];
		
        pos[atom] = selectAtomInDistribution( dBdz, Temp, &localrngState );
        
		vel[atom] = getRandomVelocity( Temp, &localrngState );
        
        acc[atom] = updateAccel( pos[atom] );
        
        isSpinUp[atom] = true;
		
		// Copy state back to global memory
		rngState[atom] = localrngState;
    }
    return;
}

__device__ double3 getRandomVelocity( double Temp, hiprandStatePhilox4_32_10_t *rngState )
{
	double3 vel = make_double3( 0., 0., 0. );
	
	double V = sqrt( d_kB*Temp/d_mRb);
	
	vel = V * getGaussianPoint( 0., 1., &rngState[0] );
    
	return vel;
}

__device__ double3 selectAtomInDistribution( double dBdz, double Temp, hiprandStatePhilox4_32_10_t *rngState )
{
    double3 r   = make_double3( 0., 0., 0. );

    double max = 0.0001;
    
    double2 r1 = ( hiprand_uniform2_double ( &rngState[0] ) * 2. - 1. ) * max;
    double  r2 = ( hiprand_uniform_double  ( &rngState[0] ) * 2. - 1. ) * max;
    
    double3 pos = make_double3( r1.x, r1.y, r2 );
    
    return pos;
}

__device__ double3 getGaussianPoint( double mean, double std, hiprandStatePhilox4_32_10_t *rngState )
{
    double2 r1 = hiprand_normal2_double ( &rngState[0] ) * std + mean;
	double  r2 = hiprand_normal_double  ( &rngState[0] ) * std + mean;
 
    double3 point = make_double3( r1.x, r1.y, r2 );
    
    return point;
}

__device__ bool pointIsInDistribution( double3 point, double dBdz, double Temp, hiprandStatePhilox4_32_10_t *rngState )
{
    bool pointIsIn = false;
    
    double potential   = 0.5*d_gs*d_muB*dBdz*sqrt( point.x*point.x + point.y*point.y + 4.*point.z*point.z );
    double probability = exp( -potential / d_kB / Temp );
    
    if ( hiprand_uniform_double ( &rngState[0] ) < probability ) {
        pointIsIn = true;
    }
    
    return pointIsIn;
}

__device__ double3 updateAccel( double3 pos )
{
    double3 accel = make_double3( 0., 0., 0. );
    
    // The rsqrt function returns the reciprocal square root of its argument
	double potential = -0.5*d_gs*d_muB*d_dBdz*rsqrt(pos.x*pos.x + pos.y*pos.y + 4.0*pos.z*pos.z)/d_mRb;
	
	accel.x =       potential * pos.x;
	accel.y =       potential * pos.y;
	accel.z = 4.0 * potential * pos.z;
    
    return accel;
}

void setInitialWavefunction( zomplex *d_psiU, zomplex *d_psiD, double2 *d_oldPops2, hbool_t *d_isSpinUp, double3 *d_pos, int numberOfAtoms )
{
    int blockSize;
	int minGridSize;
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) deviceSetInitialWavefunction,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    
    deviceSetInitialWavefunction<<<gridSize,blockSize>>>( d_psiU,
                                                          d_psiD,
                                                          d_oldPops2,
                                                          d_isSpinUp,
                                                          d_pos,
                                                          numberOfAtoms );
    
    return;
}

__global__ void deviceSetInitialWavefunction( zomplex *psiU, zomplex *psiD, double2 *oldPops2, hbool_t *isSpinUp, double3 *pos, int numberOfAtoms )
{
    for ( int atom = blockIdx.x * blockDim.x + threadIdx.x;
              atom < numberOfAtoms;
              atom += blockDim.x * gridDim.x )
    {
        double3 l_pos = pos[atom];
		double3 Bn = magneticFieldNormal( l_pos );
		
		zomplex l_psiU = 0.5 * make_hipDoubleComplex ( 1.+Bn.x+Bn.z, -Bn.y ) * rsqrt(1.+Bn.x);
		zomplex l_psiD = 0.5 * make_hipDoubleComplex ( 1.+Bn.x-Bn.z, +Bn.y ) * rsqrt(1.+Bn.x);
        
		isSpinUp[atom] = true;
		
		oldPops2[atom] = getEigenStatePops( l_psiD,
                                            l_psiU,
                                            Bn );
        
//        if (atom==0) {
//            printf("x = (%g, %g, %g), Bn = (%g, %g, %g)\n", l_pos.x, l_pos.y, l_pos.z, Bn.x, Bn.y, Bn.z );
//        }
        
        psiU[atom] = l_psiU;
        psiD[atom] = l_psiD;
	}
    
    return;
}

void initSigvrmax( double *d_sigvrmax, int numberOfCells )
{
    double sigvrmax = sqrt(3.*h_kB*Tinit/h_mRb)*8.*h_pi*h_a*h_a;
    
    cudaSetMem( d_sigvrmax, sigvrmax, numberOfCells + 1 );
}

__device__ double3 magneticField( double3 pos )
{
    double3 magneticField = make_double3( 0., 0., 0. );
    
    magneticField.x = 0.5*d_dBdz*pos.x;
    magneticField.y = 0.5*d_dBdz*pos.y;
    magneticField.z =-1.0*d_dBdz*pos.z;
    
    return magneticField;
}

__device__ double3 magneticFieldNormal( double3 pos )
{
    double3 B     = magneticField( pos );
    double  magB  = length( B );
    double3 Bn    = B / magB;
    
    return Bn;
}

__device__ double2 getEigenStatePops( zomplex psiD, zomplex psiU, double3 Bn )
{
    double2 statePopulations = make_double2( 0., 0. );
    
    // Record old populations
    statePopulations.x = 0.5 + Bn.x * ( psiD.x*psiU.x + psiD.y*psiU.y )
                             + Bn.y * ( psiD.y*psiU.x - psiD.x*psiU.y )
                             + Bn.z * ( psiU.x*psiU.x + psiU.y*psiU.y - 0.5 );
    statePopulations.y = 0.5 - Bn.x * ( psiD.x*psiU.x + psiD.y*psiU.y )
                             + Bn.y * (-psiD.y*psiU.x + psiD.x*psiU.y )
                             + Bn.z * (-psiU.x*psiU.x - psiU.y*psiU.y + 0.5 );
    
    return statePopulations;
}