#include "hip/hip_runtime.h"
//
//  setUp.cu
//  CUDADSMC
//
//  Created by Christopher Watkins on 1/08/2014.
//  Copyright (c) 2014 WIJ. All rights reserved.
//

#include "vectorMath.cuh"
#include "setUp.cuh"
#include "math.h"
#include "cudaHelpers.cuh"

#include "declareInitialSystemParameters.cuh"
#include "deviceSystemParameters.cuh"

#pragma mark - Random Number Generator
int findRNGArrayLength( void )
{
    int sizeOfRNG = 0;
    
    if (numberOfAtoms > 64*numberOfCells) {
		sizeOfRNG = numberOfAtoms;
	}
	else
	{
		sizeOfRNG = 64*numberOfCells;
	}
    
    return sizeOfRNG;
}

__global__ void initRNG( hiprandStatePhilox4_32_10_t *rngState, int numberOfAtoms )
{
	for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
		// Each thread gets the same seed, a different sequence
		// number and no offset
		hiprand_init( 1234, atom, 0, &rngState[atom] );
	}
	
	return;
}

#pragma mark - Initial Distribution

// Kernel to generate the initial distribution
__global__ void generateInitialDist(double3 *pos,
                                    double3 *vel,
                                    double3 *acc,
                                    hbool_t *isSpinUp,
                                    int     *atomID,
                                    int      numberOfAtoms,
									double   Temp,
									double   dBdz,
									hiprandStatePhilox4_32_10_t *rngState) {
    
	for (int atom = blockIdx.x * blockDim.x + threadIdx.x;
		 atom < numberOfAtoms;
		 atom += blockDim.x * gridDim.x)
	{
		/* Copy state to local memory for efficiency */
		hiprandStatePhilox4_32_10_t localrngState = rngState[atom];
		
        pos[atom] = selectAtomInDistribution( dBdz, Temp, &localrngState );
		
		vel[atom] = getRandomVelocity( Temp, &localrngState );
        
        acc[atom] = updateAccel( pos[atom] );
        
        isSpinUp[atom] = true;
        
        atomID[atom] = atom;
		
		// Copy state back to global memory
		rngState[atom] = localrngState;
    }
    return;
}

__device__ double3 getRandomVelocity( double Temp, hiprandStatePhilox4_32_10_t *rngState )
{
	double3 vel = make_double3( 0., 0., 0. );
	
	double V = sqrt(3.0*d_kB*Temp/d_mRb);
	
	vel = V * getRandomPointOnUnitSphere( &rngState[0] );
	
	return vel;
}

__device__ double3 getRandomPointOnUnitSphere( hiprandStatePhilox4_32_10_t *rngState )
{
    double2 r1 = hiprand_normal2_double ( &rngState[0] );
    double  r2 = hiprand_normal_double  ( &rngState[0] );
    
    double3 pointOnSphere = make_double3( r1.x, r1.y, r2 ) * rsqrt( r1.x*r1.x + r1.y*r1.y + r2*r2 );
    
    return pointOnSphere;
}

__device__ double3 selectAtomInDistribution( double dBdz, double Temp, hiprandStatePhilox4_32_10_t *rngState )
{
    double3 pos = make_double3( 0., 0., 0. );
    double3 r   = make_double3( 0., 0., 0. );

    double meanx = 0.0;
    double stdx  = sqrt( log( 4. ) )*d_kB*Temp / ( d_gs*d_muB*dBdz );
    
    bool noAtomSelected = true;
    
    while (noAtomSelected) {
        
        r = getGaussianPoint( meanx, stdx, &rngState[0] );
        
        if ( pointIsInDistribution( r, dBdz, Temp, &rngState[0] ) ) {
            
            pos = r;
            
            noAtomSelected = false;
        }
    }
    
    return pos;
}

__device__ double3 getGaussianPoint( double mean, double std, hiprandStatePhilox4_32_10_t *rngState )
{
    double2 r1 = hiprand_normal2_double ( &rngState[0] ) * std * 20. + mean;
	double r2  = hiprand_normal_double  ( &rngState[0] ) * std * 20. + mean;
 
    double3 point = make_double3( r1.x, r1.y, r2 );
    
    return point;
}

__device__ bool pointIsInDistribution( double3 point, double dBdz, double Temp, hiprandStatePhilox4_32_10_t *rngState )
{
    bool pointIsIn = false;
    
    double potential   = 0.5*d_gs*d_muB*dBdz*sqrt( point.x*point.x + point.y*point.y + 4.*point.z*point.z );
    double probability = exp( -potential / d_kB / Temp );
    
    if ( hiprand_uniform_double ( &rngState[0] ) < probability ) {
        pointIsIn = true;
    }
    
    return pointIsIn;
}

__device__ double3 updateAccel( double3 pos )
{
    double3 accel = make_double3( 0., 0., 0. );
    
    // The rsqrt function returns the reciprocal square root of its argument
	double potential = -0.5*d_gs*d_muB*d_dBdz*rsqrt(pos.x*pos.x + pos.y*pos.y + 4.0*pos.z*pos.z)/d_mRb;
	
	accel.x =       potential * pos.x;
	accel.y =       potential * pos.y;
	accel.z = 4.0 * potential * pos.z;
    
    return accel;
}

void setInitialWavefunction( zomplex *d_psiU, zomplex *d_psiD, double2 *d_oldPops2, hbool_t *d_isSpinUp, double3 *d_pos, int numberOfAtoms )
{
    int blockSize;
	int minGridSize;
	int gridSize;
	
	hipOccupancyMaxPotentialBlockSize( &minGridSize,
                                        &blockSize,
                                        (const void *) deviceSetInitialWavefunction,
                                        0,
                                        numberOfAtoms );
	gridSize = (numberOfAtoms + blockSize - 1) / blockSize;
    
    deviceSetInitialWavefunction<<<gridSize,blockSize>>>( d_psiU,
                                                          d_psiD,
                                                          d_oldPops2,
                                                          d_isSpinUp,
                                                          d_pos,
                                                          numberOfAtoms );
    
    return;
}

__global__ void deviceSetInitialWavefunction( zomplex *psiU, zomplex *psiD, double2 *oldPops2, hbool_t *isSpinUp, double3 *pos, int numberOfAtoms )
{
    for ( int atom = blockIdx.x * blockDim.x + threadIdx.x;
              atom < numberOfAtoms;
              atom += blockDim.x * gridDim.x )
    {
        double3 l_pos = pos[atom];
		double3 Bn = magneticFieldNormal( l_pos );
		
		zomplex l_psiU = 0.5 * make_hipDoubleComplex ( 1.+Bn.x+Bn.z, -Bn.y ) * rsqrt(1.+Bn.x);
		zomplex l_psiD = 0.5 * make_hipDoubleComplex ( 1.+Bn.x-Bn.z, +Bn.y ) * rsqrt(1.+Bn.x);
		
		isSpinUp[atom] = true;
		
		oldPops2[atom] = getEigenStatePops( l_psiD,
                                            l_psiU,
                                            Bn );
        
        psiU[atom] = l_psiU;
        psiD[atom] = l_psiD;
	}
    
    return;
}

void initSigvrmax( double *d_sigvrmax, int numberOfCells )
{
    double sigvrmax = sqrt(3.*h_kB*Tinit/h_mRb)*8.*h_pi*h_a*h_a;
    
    cudaSetMem( d_sigvrmax, sigvrmax, numberOfCells + 1 );
}

__device__ double3 magneticField( double3 pos )
{
    double3 magneticField = make_double3( 0., 0., 0. );
    
    magneticField.x = 0.5*d_dBdz*pos.x;
    magneticField.y = 0.5*d_dBdz*pos.y;
    magneticField.z =-1.0*d_dBdz*pos.z;
    
    return magneticField;
}

__device__ double3 magneticFieldNormal( double3 pos )
{
    double3 B     = magneticField( pos );
    double  magB  = length( B );
    double3 Bn    = B / magB;
    
    return Bn;
}

__device__ double2 getEigenStatePops( zomplex psiD, zomplex psiU, double3 Bn )
{
    double2 statePopulations = make_double2( 0., 0. );
    
    // Record old populations
    statePopulations.x = 0.5 + Bn.x * ( psiD.x*psiU.x + psiD.y*psiU.y )
                             + Bn.y * ( psiD.y*psiU.x - psiD.x*psiU.y )
                             + Bn.z * ( psiU.x*psiU.x + psiU.y*psiU.y - 0.5 );
    statePopulations.y = 0.5 - Bn.x * ( psiD.x*psiU.x + psiD.y*psiU.y )
                             + Bn.y * (-psiD.y*psiU.x + psiD.x*psiU.y )
                             + Bn.z * (-psiU.x*psiU.x - psiU.y*psiU.y + 0.5 );
    
    return statePopulations;
}